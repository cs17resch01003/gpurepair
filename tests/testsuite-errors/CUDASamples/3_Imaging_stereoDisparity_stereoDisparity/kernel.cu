#include "hip/hip_runtime.h"
//xfail:NOT_ALL_VERIFIED
//--gridDim=[20,67] --blockDim=[32,8]

#define blockSize_x 32
#define blockSize_y 8

// RAD is the radius of the region of support for the search
#define RAD 8
// STEPS is the number of loads we must perform to initialize the shared memory area
// (see convolution SDK sample for example)
#define STEPS 3

texture<unsigned int, hipTextureType2D, hipReadModeElementType> tex2Dleft;
texture<unsigned int, hipTextureType2D, hipReadModeElementType> tex2Dright;

__device__ static __attribute__((always_inline)) unsigned int __usad4(unsigned int A, unsigned int B, unsigned int C=0);
/* IMPERIAL EDIT: inline asm commented out
{
    unsigned int result;
#if (__CUDA_ARCH__ >= 300) // Kepler (SM 3.x) supports a 4 vector SAD SIMD
    asm("vabsdiff4.u32.u32.u32.add" " %0, %1, %2, %3;": "=r"(result):"r"(A), "r"(B), "r"(C));
#else // SM 2.0            // Fermi  (SM 2.x) supports only 1 SAD SIMD, so there are 4 instructions
    asm("vabsdiff.u32.u32.u32.add" " %0, %1.b0, %2.b0, %3;": "=r"(result):"r"(A), "r"(B), "r"(C));
    asm("vabsdiff.u32.u32.u32.add" " %0, %1.b1, %2.b1, %3;": "=r"(result):"r"(A), "r"(B), "r"(result));
    asm("vabsdiff.u32.u32.u32.add" " %0, %1.b2, %2.b2, %3;": "=r"(result):"r"(A), "r"(B), "r"(result));
    asm("vabsdiff.u32.u32.u32.add" " %0, %1.b3, %2.b3, %3;": "=r"(result):"r"(A), "r"(B), "r"(result));
#endif
    return result;
}
*/

__global__ void
stereoDisparityKernel(unsigned int *g_img0, unsigned int *g_img1,
                      unsigned int *g_odata,
                      int w, int h,
                      int minDisparity, int maxDisparity)
{
    __requires(w == 640);

    // access thread id
    const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    const int tidy = blockDim.y * blockIdx.y + threadIdx.y;
    const unsigned int sidx = threadIdx.x+RAD;
    const unsigned int sidy = threadIdx.y+RAD;

    unsigned int imLeft;
    unsigned int imRight;
    unsigned int cost;
    unsigned int bestCost = 9999999;
    unsigned int bestDisparity = 0;
    __shared__ unsigned int diff[blockSize_y+2*RAD][blockSize_x+2*RAD];

    // store needed values for left image into registers (constant indexed local vars)
    unsigned int imLeftA[STEPS];
    unsigned int imLeftB[STEPS];

    for (int i=0; i<STEPS; i++)
    {
        int offset = -RAD + i*RAD;
        imLeftA[i] = tex2D(tex2Dleft, tidx-RAD, tidy+offset);
        imLeftB[i] = tex2D(tex2Dleft, tidx-RAD+blockSize_x, tidy+offset);
    }

    // for a fixed camera system this could be hardcoded and loop unrolled
    for (int d=minDisparity; d<=maxDisparity; d++)
    {
        //LEFT
#pragma unroll
        for (int i=0;             __global_invariant(__write_implies(diff, __write_offset_bytes(diff)/sizeof(unsigned int)%(blockSize_x + 2 * RAD) == sidx - RAD)),             __global_invariant(__write_implies(diff, (__write_offset_bytes(diff)/sizeof(unsigned int)/(blockSize_x + 2 * RAD) - sidy + RAD)%RAD == 0)),             i<STEPS; i++)        
        {
            int offset = -RAD + i*RAD;
            //imLeft = tex2D( tex2Dleft, tidx-RAD, tidy+offset );
            imLeft = imLeftA[i];
            imRight = tex2D(tex2Dright, tidx-RAD+d, tidy+offset);
            cost = __usad4(imLeft, imRight);
            diff[sidy+offset][sidx-RAD] = cost;
        }

        //RIGHT
#pragma unroll

        for (int i=0;             __global_invariant(__write_implies(diff, (__write_offset_bytes(diff)/sizeof(unsigned int)%(blockSize_x + 2 * RAD) == sidx - RAD + blockSize_x)                                                    | (__write_offset_bytes(diff)/sizeof(unsigned int)%(blockSize_x + 2 * RAD) == sidx - RAD))),             __global_invariant(__write_implies(diff, (__write_offset_bytes(diff)/sizeof(unsigned int)/(blockSize_x + 2 * RAD) - sidy + RAD)%RAD == 0)),             i<STEPS; i++)        
        {
            int offset = -RAD + i*RAD;

            if (threadIdx.x < 2*RAD)
            {
                //imLeft = tex2D( tex2Dleft, tidx-RAD+blockSize_x, tidy+offset );
                imLeft = imLeftB[i];
                imRight = tex2D(tex2Dright, tidx-RAD+blockSize_x+d, tidy+offset);
                cost = __usad4(imLeft, imRight);
                diff[sidy+offset][sidx-RAD+blockSize_x] = cost;
            }
        }

        // __syncthreads();

        // sum cost horizontally
#pragma unroll

        for (int j=0; j<STEPS; j++)
        {
            int offset = -RAD + j*RAD;
            cost = 0;
#pragma unroll

            for (int i=-RAD; i<=RAD ; i++)
            {
                cost += diff[sidy+offset][sidx+i];
            }

            // __syncthreads();
            diff[sidy+offset][sidx] = cost;
            // __syncthreads();

        }

        // sum cost vertically
        cost = 0;
#pragma unroll

        for (int i=-RAD; i<=RAD ; i++)
        {
            cost += diff[sidy+i][sidx];
        }

        // see if it is better or not
        if (cost < bestCost)
        {
            bestCost = cost;
            bestDisparity = d+8;
        }

        // __syncthreads();

    }

    if (tidy < h && tidx < w)
    {
        g_odata[tidy*w + tidx] = bestDisparity;
    }
}
