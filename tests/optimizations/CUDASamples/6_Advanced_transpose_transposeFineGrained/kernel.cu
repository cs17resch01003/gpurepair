#include "hip/hip_runtime.h"
//pass
//--gridDim=[64,64] --blockDim=[16,16]

#include "common.h"

__global__ void transposeFineGrained(float *odata, float *idata, int width, int height,  int nreps)
{
    __requires(width == 1024);
    __requires(height == 1024);
    __requires(nreps == 1);

    __shared__ float block[TILE_DIM][TILE_DIM+1];

    int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
    int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
    int index = xIndex + (yIndex)*width;

    for (int r=0; r<nreps; r++)
    {
        for (int i=0; i < TILE_DIM; i += BLOCK_ROWS)
        {
            block[threadIdx.y+i][threadIdx.x] = idata[index+i*width];
        }

        __syncthreads();

        for (int i=0; i < TILE_DIM; i += BLOCK_ROWS)
        {
            odata[index+i*height] = block[threadIdx.x][threadIdx.y+i];
        }

        // IMPERIAL EDIT: add barrier
        __syncthreads();
    }
}
