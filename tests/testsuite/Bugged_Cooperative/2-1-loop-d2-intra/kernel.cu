//pass
//--blockDim=32 --gridDim=2

#include <hip/hip_runtime.h>

__global__ void test_Prog(int *A, int N) {
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  int idx = blockDim.x * bid + tid;

  for (int d = N/2; d > 0; d = d / 2) {
    for (int i = 0; i < 100; i++) {
      if (idx < d && i == 0) {
        A[idx] = A[idx + d];
      }
    }
  }
}