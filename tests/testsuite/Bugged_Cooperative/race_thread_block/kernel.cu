#include "hip/hip_runtime.h"
//pass
//--blockDim=32 --gridDim=1

#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;

__global__ void race (int* A)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  thread_block t = this_thread_block();
  int idx = blockDim.x * bid + tid;

  int temp = A[idx + 1];
  synchronize(t);
  A[idx] = temp;
}