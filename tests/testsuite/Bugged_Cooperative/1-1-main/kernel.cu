//pass
//--blockDim=32 --gridDim=2

#include <hip/hip_runtime.h>

__global__ void test_Prog(int *A, int N) {
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  int idx = blockDim.x * bid + tid;

  int tmp = A[idx + 1];
  tmp = tmp + 11;
  A[idx] += tmp;
}