#include "hip/hip_runtime.h"
//pass
//--gridDim=[64,64,1]      --blockDim=[8,8,1]

#include "common1.h"

__global__ void CUDAkernel1IDCT(float *Dst, int ImgWidth, int OffsetXBlocks, int OffsetYBlocks)
{
    __requires(ImgWidth == 512);
    // Block index
    int bx = blockIdx.x + OffsetXBlocks;
    int by = blockIdx.y + OffsetYBlocks;

    // Thread index (current image pixel)
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Texture coordinates
    const float tex_x = (float)((bx << BLOCK_SIZE_LOG2) + tx) + 0.5f;
    const float tex_y = (float)((by << BLOCK_SIZE_LOG2) + ty) + 0.5f;

    //copy current image pixel to the first block
    CurBlockLocal1[(ty << BLOCK_SIZE_LOG2) + tx ] = tex2D(TexSrc, tex_x, tex_y);

    //synchronize threads to make sure the block is copied
    __syncthreads();

    //calculate the multiplication of DCTv8matrix * A and place it in the second block
    float curelem = 0;
    int DCTv8matrixIndex = (ty << BLOCK_SIZE_LOG2) + 0;
    int CurBlockLocal1Index = 0 * BLOCK_SIZE + tx;
#pragma unroll

    for (int i=0; i<BLOCK_SIZE; i++)
    {
        curelem += DCTv8matrix[DCTv8matrixIndex] * CurBlockLocal1[CurBlockLocal1Index];
        DCTv8matrixIndex += 1;
        CurBlockLocal1Index += BLOCK_SIZE;
    }

    CurBlockLocal2[(ty << BLOCK_SIZE_LOG2) + tx ] = curelem;

    //synchronize threads to make sure the first 2 matrices are multiplied and the result is stored in the second block
    __syncthreads();

    //calculate the multiplication of (DCTv8matrix * A) * DCTv8matrixT and place it in the first block
    curelem = 0;
    int CurBlockLocal2Index = (ty << BLOCK_SIZE_LOG2) + 0;
    DCTv8matrixIndex = (tx << BLOCK_SIZE_LOG2) + 0;
#pragma unroll

    for (int i=0; i<BLOCK_SIZE; i++)
    {
        curelem += CurBlockLocal2[CurBlockLocal2Index] * DCTv8matrix[DCTv8matrixIndex];
        CurBlockLocal2Index += 1;
        DCTv8matrixIndex += 1;
    }

    CurBlockLocal1[(ty << BLOCK_SIZE_LOG2) + tx ] = curelem;

    //synchronize threads to make sure the matrices are multiplied and the result is stored back in the first block
    __syncthreads();

    //copy current coefficient to its place in the result array
    Dst[ FMUL(((by << BLOCK_SIZE_LOG2) + ty), ImgWidth) + ((bx << BLOCK_SIZE_LOG2) + tx) ] = CurBlockLocal1[(ty << BLOCK_SIZE_LOG2) + tx ];
}
