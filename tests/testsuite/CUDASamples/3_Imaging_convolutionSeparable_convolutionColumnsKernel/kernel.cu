#include "hip/hip_runtime.h"
//pass
//--gridDim=[192,48]     --blockDim=[16,8]

// in host invocation
//   assert(COLUMNS_BLOCKDIM_Y * COLUMNS_HALO_STEPS >= KERNEL_RADIUS);
//   assert(imageW % COLUMNS_BLOCKDIM_X == 0);
//   assert(imageH % (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y) == 0);

#define KERNEL_RADIUS 8
#define KERNEL_LENGTH (2 * KERNEL_RADIUS + 1)

#define   COLUMNS_BLOCKDIM_X 16
#define   COLUMNS_BLOCKDIM_Y 8
#define COLUMNS_RESULT_STEPS 8
#define   COLUMNS_HALO_STEPS 1

__constant__ float c_Kernel[KERNEL_LENGTH];

__global__ void convolutionColumnsKernel(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
)
{
    __requires(pitch == 3072);
    __shared__ float s_Data[COLUMNS_BLOCKDIM_X][(COLUMNS_RESULT_STEPS + 2 * COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + 1];

    //Offset to the upper halo edge
    const int baseX = blockIdx.x * COLUMNS_BLOCKDIM_X + threadIdx.x;
    const int baseY = (blockIdx.y * COLUMNS_RESULT_STEPS - COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + threadIdx.y;
    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;

    //Main data
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = d_Src[i * COLUMNS_BLOCKDIM_Y * pitch];
    }

    //Upper halo
#pragma unroll

    for (int i = 0; i < COLUMNS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY >= -i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
    }

    //Lower halo
#pragma unroll

    for (int i = COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS + COLUMNS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y]= (imageH - baseY > i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
    }

    //Compute and store results
    // __syncthreads();
// #pragma unroll

    for (int i = COLUMNS_HALO_STEPS;
         #define base (baseY * pitch + baseX)
         __invariant(__write_implies(d_Dst, (__write_offset_bytes(d_Dst)/sizeof(float) - base)%(COLUMNS_BLOCKDIM_Y * pitch) == 0)),
         __invariant(__write_implies(d_Dst, (__write_offset_bytes(d_Dst)/sizeof(float) - base)/(COLUMNS_BLOCKDIM_Y * pitch) >= COLUMNS_HALO_STEPS)),
         __invariant(__write_implies(d_Dst, (__write_offset_bytes(d_Dst)/sizeof(float) - base)/(COLUMNS_BLOCKDIM_Y * pitch) < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS)),
         i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
    {
        float sum = 0;
// #pragma unroll

        for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
        {
            sum += c_Kernel[KERNEL_RADIUS - j] * s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y + j];
        }

        d_Dst[i * COLUMNS_BLOCKDIM_Y * pitch] = sum;
    }
}
