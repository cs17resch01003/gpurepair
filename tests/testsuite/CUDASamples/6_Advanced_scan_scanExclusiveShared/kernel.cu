#include "hip/hip_runtime.h"
//pass
//--gridDim=6624 --blockDim=256

#include "common.h"

__device__ static __attribute__((always_inline)) uint4 scan4Inclusive(uint4 idata4, volatile uint *s_Data, uint size)
{
    //Level-0 inclusive scan
    idata4.y += idata4.x;
    idata4.z += idata4.y;
    idata4.w += idata4.z;

    //Level-1 exclusive scan
    uint oval = scan1Exclusive(idata4.w, s_Data, size / 4);

    idata4.x += oval;
    idata4.y += oval;
    idata4.z += oval;
    idata4.w += oval;

    return idata4;
}
__device__ static __attribute__((always_inline)) uint4 scan4Exclusive(uint4 idata4, volatile uint *s_Data, uint size)
{
    uint4 odata4 = scan4Inclusive(idata4, s_Data, size);
    odata4.x -= idata4.x;
    odata4.y -= idata4.y;
    odata4.z -= idata4.z;
    odata4.w -= idata4.w;
    return odata4;
}

__global__ void scanExclusiveShared(
    uint4 *d_Dst,
    uint4 *d_Src,
    uint size
)
{
    __requires((size & (size - 1)) == 0);
    __shared__ uint s_Data[2 * THREADBLOCK_SIZE];

    uint pos = blockIdx.x * blockDim.x + threadIdx.x;

    //Load data
    uint4 idata4 = d_Src[pos];

    //Calculate exclusive scan
    uint4 odata4 = scan4Exclusive(idata4, s_Data, size);

    //Write back
    d_Dst[pos] = odata4;
}
