#include "hip/hip_runtime.h"
//xfail:TIMEOUT
//--gridDim=[10,40]      --blockDim=[32,6]

template<int bx, int by> __global__ void JacobiIteration(const float *du0, const float *dv0, const float *Ix, const float *Iy, const float *Iz, int w, int h, int s, float alpha, float *du1, float *dv1);
template __global__ void JacobiIteration<32,6>(const float *du0, const float *dv0, const float *Ix, const float *Iy, const float *Iz, int w, int h, int s, float alpha, float *du1, float *dv1);
#define min(x,y) (x < y ? x : y)
#define max(x,y) (x < y ? y : x)

template<int bx, int by>
__global__
void JacobiIteration(const float *du0,
                     const float *dv0,
                     const float *Ix,
                     const float *Iy,
                     const float *Iz,
                     int w, int h, int s,
                     float alpha,
                     float *du1,
                     float *dv1)
{
    __requires(w == 320);
    __requires(h == 240);
    __requires(s == 320);
    volatile __shared__ float du[(bx + 2) * (by + 2)];
    volatile __shared__ float dv[(bx + 2) * (by + 2)];

    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;

    // position within global memory array
    const int pos = min(ix, w - 1) + min(iy, h - 1) * s;

    // position within shared memory array
    const int shMemPos = threadIdx.x + 1 + (threadIdx.y + 1) * (bx + 2);

    // Load data to shared memory.
    // load tile being processed
    du[shMemPos] = du0[pos];
    dv[shMemPos] = dv0[pos];

    // load necessary neigbouring elements
    // We clamp out-of-range coordinates.
    // It is equivalent to mirroring
    // because we access data only one step away from borders.
    if (threadIdx.y == 0)
    {
        // beginning of the tile
        const int bsx = blockIdx.x * blockDim.x;
        const int bsy = blockIdx.y * blockDim.y;
        // element position within matrix
        int x, y;
        // element position within linear array
        // gm - global memory
        // sm - shared memory
        int gmPos, smPos;

        x = min(bsx + threadIdx.x, w - 1);
        // row just below the tile
        y = max(bsy - 1, 0);
        gmPos = y * s + x;
        smPos = threadIdx.x + 1;
        du[smPos] = du0[gmPos];
        dv[smPos] = dv0[gmPos];

        // row above the tile
        y = min(bsy + by, h - 1);
        smPos += (by + 1) * (bx + 2);
        gmPos  = y * s + x;
        du[smPos] = du0[gmPos];
        dv[smPos] = dv0[gmPos];
    }
    if (threadIdx.y == 1)
    {
        // beginning of the tile
        const int bsx = blockIdx.x * blockDim.x;
        const int bsy = blockIdx.y * blockDim.y;
        // element position within matrix
        int x, y;
        // element position within linear array
        // gm - global memory
        // sm - shared memory
        int gmPos, smPos;

        y = min(bsy + threadIdx.x, h - 1);
        // column to the left
        x = max(bsx - 1, 0);
        smPos = bx + 2 + threadIdx.x * (bx + 2);
        gmPos = x + y * s;

        // check if we are within tile
        if (threadIdx.x < by)
        {
            du[smPos] = du0[gmPos];
            dv[smPos] = dv0[gmPos];
            // column to the right
            x = min(bsx + bx, w - 1);
            gmPos  = y * s + x;
            smPos += bx + 1;
            du[smPos] = du0[gmPos];
            dv[smPos] = dv0[gmPos];
        }
    }

    // __syncthreads();

    if (ix >= w || iy >= h)
    {
     return;
    }

    // now all necessary data are loaded to shared memory
    int left, right, up, down;
    left  = shMemPos - 1;
    right = shMemPos + 1;
    up    = shMemPos + bx + 2;
    down  = shMemPos - bx - 2;

    float sumU = (du[left] + du[right] + du[up] + du[down]) * 0.25f;
    float sumV = (dv[left] + dv[right] + dv[up] + dv[down]) * 0.25f;

    float frac = (Ix[pos] * sumU + Iy[pos] * sumV + Iz[pos])
                 / (Ix[pos] * Ix[pos] + Iy[pos] * Iy[pos] + alpha);

    du1[pos] = sumU - Ix[pos] * frac;
    dv1[pos] = sumV - Iy[pos] * frac;
}
