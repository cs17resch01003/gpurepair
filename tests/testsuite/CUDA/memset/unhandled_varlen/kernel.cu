
#include <hip/hip_runtime.h>
//xfail:BUGLE_ERROR
//--gridDim=1 --blockDim=32 --no-inline

//This kernel is racy: memset is called with variable length.
#define memset(dst,val,len) __builtin_memset(dst,val,len)

__device__ int bar(void);

__global__ void kernel(uint4 *out) {
  uint4 vector;
  int len = bar();
  memset(&vector, 0, len);
  out[threadIdx.x] = vector;
}
