#include "hip/hip_runtime.h"
//pass
//--blockDim=64 --gridDim=1 --no-inline

#include "hip/hip_runtime.h"


__device__ int* bar(int* p) {
  __ensures(__implies(__enabled(), __return_val_ptr() == p));
  return p;
}

__global__ void foo(int* p) {

  int* q = bar(p);

  q[threadIdx.x] = 0;

}
