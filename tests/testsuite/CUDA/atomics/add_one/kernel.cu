//pass
//--blockDim=2 --gridDim=1 --no-inline

#include <hip/hip_runtime.h>

__global__ void race_test (unsigned int* i, int* A)
{
  int tid = threadIdx.x;
  int j = atomicAdd(i,1);
  A[j] = tid;
}
