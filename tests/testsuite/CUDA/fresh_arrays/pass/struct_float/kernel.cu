#include "hip/hip_runtime.h"
//pass
//--blockDim=2048 --gridDim=64

struct s {
  float *p;
};

__global__ void foo(s q) {
  __requires_fresh_array(q.p);
  q.p[threadIdx.x + blockIdx.x * blockDim.x] = 4.2f;
}
