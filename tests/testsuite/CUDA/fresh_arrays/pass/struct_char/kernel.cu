#include "hip/hip_runtime.h"
//pass
//--blockDim=2048 --gridDim=64

struct s {
  char *p;
};

__global__ void foo(s q) {
  __requires_fresh_array(q.p);
  q.p[threadIdx.x + blockIdx.x * blockDim.x] = 42;
}
