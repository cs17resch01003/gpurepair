//pass
//--blockDim=512 --gridDim=64 --loop-unwind=2 --no-inline

#include <hip/hip_runtime.h>

extern "C" {

__global__ void helloCUDA(float *A)
{
    __shared__ float B[256];
    for(int i = 0; i < 10; i ++) {
        B[i] = A[i];
    }

}

}
