//pass
//--blockDim=32 --gridDim=1

#include <hip/hip_runtime.h>

__global__ void test_Prog(int *A, int N) { 
	const int tid = threadIdx.x;

	int alpha=A[tid+1];
	if(tid>=0)
	{
		int temp2=A[tid+2];
		__syncthreads();
		A[tid]+=temp2;
	}
	A[tid]+=alpha;
}
	