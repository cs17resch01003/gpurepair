//pass
//--blockDim=32 --gridDim=1

#include <hip/hip_runtime.h>

__global__ void race (int* A)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  int idx = blockDim.x * bid + tid;

  int temp = 10;

  A[idx] = temp;
  temp = A[idx + 1];
}