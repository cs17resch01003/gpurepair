//pass
//--blockDim=32 --gridDim=1

#include <hip/hip_runtime.h>

__global__ void race (int* A)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  int idx = blockDim.x * bid + tid;

  int temp = A[idx + 1];
  __syncthreads();
  A[idx] = temp;
}