
#include <hip/hip_runtime.h>
//xfail:BUGLE_ERROR
//--gridDim=1 --blockDim=32 --no-inline

//This kernel is not-racy: memset is called with variable value.
#define memset(dst,val,len) __builtin_memset(dst,val,len)

__device__ int bar(void);

__global__ void kernel(uint4 *out) {
  uint4 vector;
  int val = bar();
  memset(&vector, val, 16);
  out[threadIdx.x] = vector;
}
