#include "hip/hip_runtime.h"
//pass
//--blockDim=512 --gridDim=1 --no-inline

#include <hip/hip_runtime.h>

__global__ void curand_test(hiprandState *state, float *A) {
   A[threadIdx.x] = hiprand(&state[threadIdx.x]);
}
