//pass
//--blockDim=256 --gridDim=1 --no-inline

#include <hip/hip_runtime.h>

__global__ void curand_test(hiprandStateMtgp32_t *state, float *A) {
   A[threadIdx.x] = hiprand(state);
}
