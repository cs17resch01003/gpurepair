#include "hip/hip_runtime.h"
//pass
//--gridDim=64               --blockDim=256

#include "common.h"

#define MERGE_THREADBLOCK_SIZE 256

__global__ void mergeHistogram64Kernel(
    uint *d_Histogram,
    uint *d_PartialHistograms,
    uint histogramCount
)
{
    __shared__ uint data[MERGE_THREADBLOCK_SIZE];

    uint sum = 0;

    for (uint i = threadIdx.x; i < histogramCount; i += MERGE_THREADBLOCK_SIZE)
    {
        sum += d_PartialHistograms[blockIdx.x + i * HISTOGRAM64_BIN_COUNT];
    }

    data[threadIdx.x] = sum;

    for (uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1)
    {
        // __syncthreads();

        if (threadIdx.x < stride)
        {
            data[threadIdx.x] += data[threadIdx.x + stride];
        }
    }

    if (threadIdx.x == 0)
    {
        d_Histogram[blockIdx.x] = data[0];
    }
}
