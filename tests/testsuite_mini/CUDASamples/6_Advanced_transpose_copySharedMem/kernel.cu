#include "hip/hip_runtime.h"
//pass
//--gridDim=[64,64] --blockDim=[16,16]

#include "common.h"

__global__ void copySharedMem(float *odata, float *idata, int width, int height, int nreps)
{
    __requires(width == 1024);
    __requires(height == 1024);
    __requires(nreps == 1);

    __shared__ float tile[TILE_DIM][TILE_DIM];

    int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
    int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;

    int index  = xIndex + width*yIndex;

    for (int r=0; r < nreps; r++)
    {
        for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS)
        {
            if (xIndex < width && yIndex < height)
            {
                tile[threadIdx.y][threadIdx.x] = idata[index];
            }
        }

        __syncthreads();

        for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS)
        {
            if (xIndex < height && yIndex < width)
            {
                odata[index] = tile[threadIdx.y][threadIdx.x];
            }
        }
    }
}
