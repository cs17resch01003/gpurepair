//pass
//--blockDim=32 --gridDim=1

#include <hip/hip_runtime.h>

__global__ void test_Prog(int *A,int *B, int N) { 
	const int tid = threadIdx.x;
	int tmp=A[tid+1];
	int tmp2=B[tid+1];
__syncthreads();
	B[tid]=tmp2+tmp;
	A[tid]=tmp2-tmp;	
}