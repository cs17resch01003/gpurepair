
#include <hip/hip_runtime.h>
//pass
//--gridDim=256 --blockDim=256

__global__ void shfl_scan_test(int *data, int width, int *partial_sums=NULL)
{
    extern __shared__ int sums[];
    int id = ((blockIdx.x * blockDim.x) + threadIdx.x);
    int lane_id = id % warpSize;
    // determine a warp_id within a block
    int warp_id = threadIdx.x / warpSize;

    // Below is the basic structure of using a shfl instruction
    // for a scan.
    // Record "value" as a variable - we accumulate it along the way
    int value = data[id];

    // Now accumulate in log steps up the chain
    // compute sums, with another thread's value who is
    // distance delta away (i).  Note
    // those threads where the thread 'i' away would have
    // been out of bounds of the warp are unaffected.  This
    // creates the scan sum.
#pragma unroll

    for (int i=1; i<=width; i*=2)
    {
        int n = __shfl_up(value, i, width);

        if (lane_id >= i){
         value += n;
        }
    }

    // value now holds the scan value for the individual thread
    // next sum the largest values for each warp

    // write the sum of the warp to smem
    if (threadIdx.x % warpSize == warpSize-1)
    {
        sums[warp_id] = value;
    }

__syncthreads();
    // __syncthreads();
// 
    //
    // scan sum the warp sums
    // the same shfl scan operation, but performed on warp sums
    //
    if (warp_id == 0)
    {
        int warp_sum = sums[lane_id];

        for (int i=1; i<=width; i*=2)
        {
            int n = __shfl_up(warp_sum, i, width);

            if (lane_id >= i){
             warp_sum += n;
            }
        }

        sums[lane_id] = warp_sum;
    }

    // __syncthreads();

    // perform a uniform add across warps in the block
    // read neighbouring warp's sum and add it to threads value
    int blockSum = 0;

__syncthreads();
    if (warp_id > 0)
    {
        blockSum = sums[warp_id-1];
    }

    value += blockSum;

    // Now write out our result
    data[id] = value;

    // last thread has sum, write write out the block's sum
    if (partial_sums != NULL && threadIdx.x == blockDim.x-1)
    {
        partial_sums[blockIdx.x] = value;
    }
}
