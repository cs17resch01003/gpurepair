#include "hip/hip_runtime.h"
//pass
//--gridDim=[32,32] --blockDim=[16,16]

typedef unsigned int uint;
#define BLOCK_DIM 16

__global__ void d_transpose(uint *odata, uint *idata, int width, int height)
{
    __requires(width == 512);
    __requires(height == 512);
    __shared__ uint block[BLOCK_DIM][BLOCK_DIM+1];

    // read the matrix tile into shared memory
    unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
    unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;

    if ((xIndex < width) && (yIndex < height))
    {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = idata[index_in];
__syncthreads();
    }

    // __syncthreads();

    // write the transposed matrix tile to global memory
    xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
    yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;

    if ((xIndex < height) && (yIndex < width))
    {
        unsigned int index_out = yIndex * height + xIndex;
        odata[index_out] = block[threadIdx.x][threadIdx.y];
    }
}
