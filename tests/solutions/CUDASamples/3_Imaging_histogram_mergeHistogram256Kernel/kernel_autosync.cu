#include "hip/hip_runtime.h"
//pass
//--gridDim=256              --blockDim=256

#include "common.h"

#define MERGE_THREADBLOCK_SIZE 256

__global__ void mergeHistogram256Kernel(
    uint *d_Histogram,
    uint *d_PartialHistograms,
    uint histogramCount
)
{
    uint sum = 0;

    for (uint i = threadIdx.x; i < histogramCount; i += MERGE_THREADBLOCK_SIZE)
    {
        sum += d_PartialHistograms[blockIdx.x + i * HISTOGRAM256_BIN_COUNT];
    }

    __shared__ uint data[MERGE_THREADBLOCK_SIZE];
    data[threadIdx.x] = sum;

    for (uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1)
    {
        // __syncthreads();
__syncthreads();

        if (threadIdx.x < stride)
        {
            data[threadIdx.x] += data[threadIdx.x + stride];
        }
    }

    if (threadIdx.x == 0)
    {
        d_Histogram[blockIdx.x] = data[0];
    }
}
