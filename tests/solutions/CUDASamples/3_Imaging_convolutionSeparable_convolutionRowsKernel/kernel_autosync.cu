#include "hip/hip_runtime.h"
//pass
//--gridDim=[24,768]     --blockDim=[16,4]

// in host invocation
//   assert(ROWS_BLOCKDIM_X * ROWS_HALO_STEPS >= KERNEL_RADIUS);
//   assert(imageW % (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X) == 0);
//   assert(imageH % ROWS_BLOCKDIM_Y == 0);

#define KERNEL_RADIUS 8
#define KERNEL_LENGTH (2 * KERNEL_RADIUS + 1)

#define   ROWS_BLOCKDIM_X 16
#define   ROWS_BLOCKDIM_Y 4
#define ROWS_RESULT_STEPS 8
#define   ROWS_HALO_STEPS 1

__constant__ float c_Kernel[KERNEL_LENGTH];

__global__ void convolutionRowsKernel(
    float *d_Dst,
    float *d_Src,
    int imageW,
    int imageH,
    int pitch
)
{
    __requires(pitch == 3072);
    __shared__ float s_Data[ROWS_BLOCKDIM_Y][(ROWS_RESULT_STEPS + 2 * ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X];

    //Offset to the left halo edge
    const int baseX = (blockIdx.x * ROWS_RESULT_STEPS - ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X + threadIdx.x;
    const int baseY = blockIdx.y * ROWS_BLOCKDIM_Y + threadIdx.y;

    d_Src += baseY * pitch + baseX;
    d_Dst += baseY * pitch + baseX;

    //Load main data
#pragma unroll

    for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = d_Src[i * ROWS_BLOCKDIM_X];
    }

    //Load left halo
#pragma unroll

    for (int i = 0; i < ROWS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX >= -i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
    }

    //Load right halo
// #pragma unroll

    for (int i = ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS + ROWS_HALO_STEPS; i++)
    {
        s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (imageW - baseX > i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
    }

    //Compute and store results
__syncthreads();
    // __syncthreads();
// #pragma unroll

    for (int i = ROWS_HALO_STEPS;
         #define base (baseY * pitch + baseX)
         __invariant(__write_implies(d_Dst, (__write_offset_bytes(d_Dst)/sizeof(float) - base)%(ROWS_BLOCKDIM_X) == 0)),
         __invariant(__write_implies(d_Dst, (__write_offset_bytes(d_Dst)/sizeof(float) - base)/(ROWS_BLOCKDIM_X) >= ROWS_HALO_STEPS)),
         __invariant(__write_implies(d_Dst, (__write_offset_bytes(d_Dst)/sizeof(float) - base)/(ROWS_BLOCKDIM_X) < ROWS_HALO_STEPS + ROWS_RESULT_STEPS)),
         i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
    {
        float sum = 0;

#pragma unroll

        for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
        {
            sum += c_Kernel[KERNEL_RADIUS - j] * s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X + j];
        }

        d_Dst[i * ROWS_BLOCKDIM_X] = sum;
    }
}
