#include "hip/hip_runtime.h"
//pass
//--gridDim=1 --blockDim=512 --no-inline

__global__ void example(float * A, int x) {
    __requires(x >= 0);
    __requires(x < 1000);

    if(threadIdx.x == 15) {
        A[threadIdx.x + x] = threadIdx.x;
    }

__syncthreads();
    if(threadIdx.x == 200) {
        A[threadIdx.x] = threadIdx.x;
    }
}
