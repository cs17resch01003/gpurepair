#include "hip/hip_runtime.h"
//pass
//--gridDim=[64,64,1]      --blockDim=[8,8,1]

#include "common_quantization.h"
__global__ void CUDAkernelQuantizationFloat(float *SrcDst, int Stride)
{
    __requires(Stride == 512);
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index (current coefficient)
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    //copy current coefficient to the local variable
    float curCoef = SrcDst[(by * BLOCK_SIZE + ty) * Stride + (bx * BLOCK_SIZE + tx) ];
    float curQuant = (float)Q[ ty * BLOCK_SIZE + tx ];

    //quantize the current coefficient
    float quantized = round(curCoef / curQuant);
    curCoef = quantized * curQuant;

    //copy quantized coefficient back to the DCT-plane
    SrcDst[(by * BLOCK_SIZE + ty) * Stride + (bx * BLOCK_SIZE + tx) ] = curCoef;
}
