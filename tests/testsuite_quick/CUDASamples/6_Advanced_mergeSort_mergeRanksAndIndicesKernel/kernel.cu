#include "hip/hip_runtime.h"
//pass
//--gridDim=64               --blockDim=256

#include "common_merge.h"

__global__ void mergeRanksAndIndicesKernel(
    uint *d_Limits,
    uint *d_Ranks,
    uint stride,
    uint N,
    uint threadCount
)
{
    __requires(stride & (stride - 1) == 0);
    __requires(stride < N);
    uint pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos >= threadCount)
    {
        return;
    }

    const uint           i = pos & ((stride / SAMPLE_STRIDE) - 1);
    const uint segmentBase = (pos - i) * (2 * SAMPLE_STRIDE);
    d_Ranks  += (pos - i) * 2;
    d_Limits += (pos - i) * 2;

    const uint segmentElementsA = stride;
    const uint segmentElementsB = umin(stride, N - segmentBase - stride);
    const uint  segmentSamplesA = getSampleCount(segmentElementsA);
    const uint  segmentSamplesB = getSampleCount(segmentElementsB);

    if (i < segmentSamplesA)
    {
        uint dstPos = binarySearchExclusive<1U>(d_Ranks[i], d_Ranks + segmentSamplesA, segmentSamplesB, nextPowerOfTwo(segmentSamplesB)) + i;
        d_Limits[dstPos] = d_Ranks[i];
    }

    if (i < segmentSamplesB)
    {
        uint dstPos = binarySearchInclusive<1U>(d_Ranks[segmentSamplesA + i], d_Ranks, segmentSamplesA, nextPowerOfTwo(segmentSamplesA)) + i;
        d_Limits[dstPos] = d_Ranks[segmentSamplesA + i];
    }
}
