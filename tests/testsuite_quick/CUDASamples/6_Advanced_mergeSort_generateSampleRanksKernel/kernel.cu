#include "hip/hip_runtime.h"
//pass
//--gridDim=64               --blockDim=256

#include "common_merge.h"

template<uint sortDir> __global__ void generateSampleRanksKernel(uint *d_RanksA, uint *d_RanksB, uint *d_SrcKey, uint stride, uint N, uint threadCount);
template               __global__ void generateSampleRanksKernel<1>(uint *d_RanksA, uint *d_RanksB, uint *d_SrcKey, uint stride, uint N, uint threadCount);

template<uint sortDir> __global__ void generateSampleRanksKernel(
    uint *d_RanksA,
    uint *d_RanksB,
    uint *d_SrcKey,
    uint stride,
    uint N,
    uint threadCount
)
{
    __requires(stride & (stride - 1) == 0);
    uint pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos >= threadCount)
    {
        return;
    }

    const uint           i = pos & ((stride / SAMPLE_STRIDE) - 1);
    const uint segmentBase = (pos - i) * (2 * SAMPLE_STRIDE);
    d_SrcKey += segmentBase;
    d_RanksA += segmentBase / SAMPLE_STRIDE;
    d_RanksB += segmentBase / SAMPLE_STRIDE;

    const uint segmentElementsA = stride;
    const uint segmentElementsB = umin(stride, N - segmentBase - stride);
    const uint  segmentSamplesA = getSampleCount(segmentElementsA);
    const uint  segmentSamplesB = getSampleCount(segmentElementsB);

    if (i < segmentSamplesA)
    {
        d_RanksA[i] = i * SAMPLE_STRIDE;
        d_RanksB[i] = binarySearchExclusive<sortDir>(
                          d_SrcKey[i * SAMPLE_STRIDE], d_SrcKey + stride,
                          segmentElementsB, nextPowerOfTwo(segmentElementsB)
                      );
    }

    if (i < segmentSamplesB)
    {
        d_RanksB[(stride / SAMPLE_STRIDE) + i] = i * SAMPLE_STRIDE;
        d_RanksA[(stride / SAMPLE_STRIDE) + i] = binarySearchInclusive<sortDir>(
                                                     d_SrcKey[stride + i * SAMPLE_STRIDE], d_SrcKey + 0,
                                                     segmentElementsA, nextPowerOfTwo(segmentElementsA)
                                                 );
    }
}
