#include "hip/hip_runtime.h"
//pass
//--gridDim=195 --blockDim=128

__global__ void initRNG(hiprandState *const rngStates,
                        const unsigned int seed)
{
    // Determine thread ID
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialise the RNG
    hiprand_init(seed, tid, 0, &rngStates[tid]);
}
