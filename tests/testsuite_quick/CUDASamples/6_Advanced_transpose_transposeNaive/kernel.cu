#include "hip/hip_runtime.h"
//pass
//--gridDim=[64,64] --blockDim=[16,16]

#include "common.h"

__global__ void transposeNaive(float *odata, float *idata, int width, int height, int nreps)
{
    __requires(height == 1024);

    int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
    int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;

    int index_in  = xIndex + width * yIndex;
    int index_out = yIndex + height * xIndex;

    for (int r=0; r < nreps; r++)
    {
        for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS)
        {
            odata[index_out+i] = idata[index_in+i*width];
        }
    }
}
