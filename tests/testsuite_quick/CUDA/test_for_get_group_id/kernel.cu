//pass
//--blockDim=64 --gridDim=1 --no-inline

#include "hip/hip_runtime.h"


__global__ void foo(float* A) {

  if(blockIdx.x == 0) {
    A[threadIdx.x] = 42.f;
  }

}
