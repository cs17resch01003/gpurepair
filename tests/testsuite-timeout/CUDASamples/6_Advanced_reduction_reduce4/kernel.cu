#include "hip/hip_runtime.h"
//xfail:TIMEOUT
//--gridDim=64 --blockDim=256 --warp-sync=32

template <class T, unsigned int blockSize> __global__ void reduce4(T *g_idata, T *g_odata, unsigned int n);
template __global__ void reduce4<int,256>(int *g_idata, int *g_odata, unsigned int n);

#include "common.h"

template <class T, unsigned int blockSize>
__global__ void
reduce4(T *g_idata, T *g_odata, unsigned int n)
{
    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

   // T mySum = (i < n) ? g_idata[i] : 0;
   T mySum=0;
    if (i<n){
        T mySum = g_idata[i];
    }



    if (i + blockSize < n){
        mySum += g_idata[i+blockSize];
    }

    sdata[tid] = mySum;
    // __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>32; s>>=1)
    {
        if (tid < s)
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }

        // __syncthreads();
    }

    if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile T *smem = sdata;

        if (blockSize >=  64)
        {
            smem[tid] = mySum = mySum + smem[tid + 32];
        }

        if (blockSize >=  32)
        {
            smem[tid] = mySum = mySum + smem[tid + 16];
        }

        if (blockSize >=  16)
        {
            smem[tid] = mySum = mySum + smem[tid +  8];
        }

        if (blockSize >=   8)
        {
            smem[tid] = mySum = mySum + smem[tid +  4];
        }

        if (blockSize >=   4)
        {
            smem[tid] = mySum = mySum + smem[tid +  2];
        }

        if (blockSize >=   2)
        {
            smem[tid] = mySum = mySum + smem[tid +  1];
        }
    }

    // write result for this block to global mem
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}
