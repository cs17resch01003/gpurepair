//pass
//--blockDim=32 --gridDim=1 --no-inline

#include <hip/hip_runtime.h>

__global__ void race (int* __restrict__ A1, int* __restrict__ A2, int* __restrict__ A3, int* __restrict__ A4)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  int idx = blockDim.x * bid + tid;
  int temp = 0;

  temp = A1[idx + 1];
  A1[idx] = temp;

  temp = A2[idx + 1];
  A2[idx] = temp;

  temp = A3[idx + 1];
  A3[idx] = temp;

  temp = A4[idx + 1];
  A4[idx] = temp;
}