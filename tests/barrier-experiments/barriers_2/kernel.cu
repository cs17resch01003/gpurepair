//pass
//--blockDim=32 --gridDim=1

#include <hip/hip_runtime.h>

__global__ void race (int* __restrict__ A1, int* __restrict__ A2)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  int idx = blockDim.x * bid + tid;
  int temp = 0;

  temp = A1[idx + 1];
  A1[idx] = temp;

  temp = A2[idx + 1];
  A2[idx] = temp;
}