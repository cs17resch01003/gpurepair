//pass
//--blockDim=32 --gridDim=1

#include <hip/hip_runtime.h>

__global__ void test_Prog(int *A, int N) { 
	const int tid = threadIdx.x;
	for(int d = N/2; d > 0; d = d / 2) {
	  if (tid < d) {
	    A[tid] += A[tid + d];
	  }
	}
}
