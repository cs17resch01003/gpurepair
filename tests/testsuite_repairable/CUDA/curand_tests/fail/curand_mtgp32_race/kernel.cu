//xfail:REPAIR_ERROR
//--blockDim=512 --gridDim=1 --no-inline

#include <hip/hip_runtime.h>

__global__ void curand_test(hiprandStateMtgp32_t *state, float *A) {
   A[threadIdx.x] = hiprand(state);
}
