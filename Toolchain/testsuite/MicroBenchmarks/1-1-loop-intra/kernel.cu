//xfail:NOT_ALL_VERIFIED
//--blockDim=32 --gridDim=1 --no-inline

#include <hip/hip_runtime.h>

__global__ void test_Prog(int *A, int N) { 
	const int tid = threadIdx.x;
	for(int d = N/2; d > 0; d = d / 2) {
	  if (tid < d) {
	    A[tid] += A[tid + d];
	  }
	}
}
