
#include <hip/hip_runtime.h>
//xfail:NOT_ALL_VERIFIED
//--blockDim=1024 --gridDim=1 --no-inline
//possible attempt to modify constant memory

__constant__ int A[1024];

__global__ void foo(int *B) {
  A[threadIdx.x] = B[threadIdx.x];
}
