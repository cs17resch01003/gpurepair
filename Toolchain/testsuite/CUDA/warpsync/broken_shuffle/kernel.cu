//xfail:NOT_ALL_VERIFIED
//--blockDim=1024 --gridDim=1 --warp-sync=16 --no-inline

#include <hip/hip_runtime.h>

__global__ void shuffle (int* A)
{
	int tid = threadIdx.x;
	int warp = tid / 32;
	int* B = A + (warp*32);
	A[tid] = B[(tid + 1)%32];
}
