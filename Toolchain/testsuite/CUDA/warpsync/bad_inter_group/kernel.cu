
#include <hip/hip_runtime.h>
//xfail:NOT_ALL_VERIFIED
//--blockDim=128 --gridDim=128 --warp-sync=32 --no-inline
//kernel.cu: error: possible read-write race on A

__global__ void foo(int* A) {

    A[ blockIdx.x*blockDim.x + threadIdx.x ] += (A[ (blockIdx.x + 1)*blockDim.x + threadIdx.x ]);

}
