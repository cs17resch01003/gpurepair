//xfail:NOT_ALL_VERIFIED
//--blockDim=512 --gridDim=64 --loop-unwind=2 --no-inline
//kernel.cu: error: possible write-write race on B
#include <hip/hip_runtime.h>

extern "C" {

__global__ void helloCUDA(float *A)
{
    __shared__ float B[256];
    for(int i = 0; i < 10; i ++) {
        B[i] = A[i];
    }

}

}
