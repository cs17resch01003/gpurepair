#include "hip/hip_runtime.h"
//xfail:NOT_ALL_VERIFIED
//--blockDim=2048 --gridDim=64
//possible write-write race on q.p\[0\]

struct s {
  char *p;
};

__global__ void foo(s q) {
  __requires_fresh_array(q.p);
  q.p[0] = threadIdx.x;
}
