#include "hip/hip_runtime.h"
//xfail:NOT_ALL_VERIFIED
//--blockDim=2048 --gridDim=64
//possible write-write race on q.v.p\[0\]

struct s {
  float *p;
};

struct t {
  s v;
};

__global__ void foo(t q) {
  __requires_fresh_array(q.v.p);
  q.v.p[0] = threadIdx.x;
}
