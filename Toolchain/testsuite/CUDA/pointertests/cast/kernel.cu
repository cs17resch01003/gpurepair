//xfail:NOT_ALL_VERIFIED
//--blockDim=32 --gridDim=64 --no-inline
//error: possible write-write race on


#include "hip/hip_runtime.h"

#define N 32


__global__ void foo(int* p) {
  __shared__ unsigned char x[N];

  for (unsigned int i=0; i<(N/4); i++) {
    ((unsigned int *)x)[i] = threadIdx.x;
  }
}
