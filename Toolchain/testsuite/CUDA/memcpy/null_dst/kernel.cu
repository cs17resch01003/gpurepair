
#include <hip/hip_runtime.h>
//xfail:NOT_ALL_VERIFIED
//--gridDim=1 --blockDim=2 --no-inline

//This kernel has a null pointer access.

#define memcpy(dst, src, len) __builtin_memcpy(dst, src, len)

typedef struct {
  short x;
  short y;
} s_t; //< sizeof(s2_t) == 4

__global__ void k(s_t *in, s_t *out) {
  memcpy(0, &in[threadIdx.x], sizeof(s_t));
}
