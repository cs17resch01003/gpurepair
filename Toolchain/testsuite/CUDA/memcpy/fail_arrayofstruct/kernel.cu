
#include <hip/hip_runtime.h>
//xfail:NOT_ALL_VERIFIED
//--gridDim=1 --blockDim=2 --no-inline

//This kernel is racy.
//
//It uses uses struct-assignment, which is translated into a memcpy by clang and
//dealt with as a series of reads/writes by bugle.

typedef struct {
  short x;
  short y;
} pair_t;

__global__ void k(pair_t *pairs) {
  pair_t fresh;
  pairs[42] = fresh;
}
