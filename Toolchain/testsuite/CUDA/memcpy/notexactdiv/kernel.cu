
#include <hip/hip_runtime.h>
//xfail:NOT_ALL_VERIFIED
//--gridDim=1 --blockDim=2 --no-inline

//This kernel is racy.
//
//The memcpy destination is unaligned so we have to handle the arrays in and out
//at the byte-level.

#define memcpy(dst, src, len) __builtin_memcpy(dst, src, len)

typedef struct {
  short x;
  short y;
} s_t; //< sizeof(s2_t) == 4

__global__ void k(s_t *in, s_t *out) {
  size_t len = 6;
  memcpy((char*)&out[threadIdx.x]+1, &in[threadIdx.x], len);
}
