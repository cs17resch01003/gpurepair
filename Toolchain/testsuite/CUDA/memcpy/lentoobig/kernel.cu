
#include <hip/hip_runtime.h>
//xfail:NOT_ALL_VERIFIED
//--gridDim=1 --blockDim=2 --no-inline

//This kernel is racy.
//
//The memcpy resolves to a non-integer number of element writes so we have to
//handle the arrays in and out at the byte-level.

#define memcpy(dst, src, len) __builtin_memcpy(dst, src, len)

typedef struct {
  short x;
  short y;
} s_t; //< sizeof(s_t) == 4

__global__ void k(s_t *in, s_t *out) {
  size_t len = 5;
  memcpy(&out[threadIdx.x], &in[threadIdx.x], len);
}
