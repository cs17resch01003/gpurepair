#include "hip/hip_runtime.h"
//xfail:REPAIR_ERROR
//--blockDim=2 --gridDim=1 --no-inline

#include <hip/hip_runtime.h>

__global__ void curand_test(hiprandState *state, float *A) {
   A[threadIdx.x] = hiprand_uniform(state);
}
