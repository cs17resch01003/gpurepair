#include "hip/hip_runtime.h"
//xfail:NOT_ALL_VERIFIED
//--blockDim=2 --gridDim=1 --no-inline
//Write by thread .+kernel\.cu:8:21:

#include <hip/hip_runtime.h>

__global__ void curand_test(hiprandState *state, float *A) {
   A[threadIdx.x] = hiprand_uniform(state);
}
