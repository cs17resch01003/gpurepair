//xfail:REPAIR_ERROR
//--blockDim=256 --gridDim=2 --no-inline

#include <hip/hip_runtime.h>

__global__ void curand_test(hiprandStateMtgp32_t *state, float *A) {
  if (threadIdx.x == 0) {
    A[blockIdx.x] = hiprand(state);
  }
}
