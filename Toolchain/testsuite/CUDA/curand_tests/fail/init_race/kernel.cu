#include "hip/hip_runtime.h"
//xfail:NOT_ALL_VERIFIED
//--blockDim=2 --gridDim=1 --no-inline
//Write by thread .+kernel.cu:8:4:

#include <hip/hip_runtime.h>

__global__ void init_test(hiprandState *state, unsigned int *A) {
   hiprand_init(0, 0, 0, state);

   __syncthreads();

   if (threadIdx.x == 0) {
     A[0] = hiprand(state);
   }
}
