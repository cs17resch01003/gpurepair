//xfail:NOT_ALL_VERIFIED
//--blockDim=512 --gridDim=1 --no-inline
//Write by thread .+kernel\.cu:8:21:

#include <hip/hip_runtime.h>

__global__ void curand_test(hiprandStateMtgp32_t *state, float *A) {
   A[threadIdx.x] = hiprand(state);
}
