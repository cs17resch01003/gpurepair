
#include <hip/hip_runtime.h>
//xfail:NOT_ALL_VERIFIED
//--blockDim=1024 --gridDim=1 --no-inline
//error: possible null pointer access

__device__ float multiplyByTwo(float *v, unsigned int tid)
{
    return v[tid] * 2.0f;
}

__device__ float divideByTwo(float *v, unsigned int tid)
{
    return v[tid] * 0.5f;
}

typedef float(*funcType)(float*, unsigned int);

__global__ void foor(float *v, unsigned int size, unsigned int i)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    funcType f;

    if (i == 1)
      f = multiplyByTwo;
    else if (i == 2)
      f = divideByTwo;
    else
      f = NULL;

    if (tid < size)
    {
        float x = (*f)(v, tid);
        x += multiplyByTwo(v, tid);
    }
}
