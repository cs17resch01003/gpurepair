#include "hip/hip_runtime.h"
//pass
//--blockDim=1024 --gridDim=1 --no-inline

__device__ float multiplyByTwo(float *v, unsigned int tid)
{
    return v[tid] * 2.0f;
}

__device__ float divideByTwo(float *v, unsigned int tid)
{
    return v[tid] * 0.5f;
}

typedef float(*funcType)(float*, unsigned int);

__global__ void foor(float *v, unsigned int size, unsigned int i)
{
    __requires(i == 1 | i == 2);
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    funcType f;

    if (i == 1)
      f = multiplyByTwo;
    else if (i == 2)
      f = divideByTwo;
    else
      f = NULL;

    if (tid < size)
    {
        float x = (*f)(v, tid);
        x += multiplyByTwo(v, tid);
    }
}
