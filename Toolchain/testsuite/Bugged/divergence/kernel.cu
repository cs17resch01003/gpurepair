#include <hip/hip_runtime.h>

__global__ void race (int* A)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  int idx = blockDim.x * bid + tid;

  if (idx % 2 == 0)
  {
	  int temp = A[idx + 2];
	  A[idx] = temp;
  }
}