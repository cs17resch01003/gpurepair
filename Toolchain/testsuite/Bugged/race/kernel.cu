//pass
//--blockDim=32 --gridDim=1 --no-inline

#include <hip/hip_runtime.h>

__global__ void race (int* A)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  int idx = blockDim.x * bid + tid;

  int temp = A[idx + 1];
  A[idx] = temp;
}