#include "hip/hip_runtime.h"
//pass
//--gridDim=1024             --blockDim=512

#include "common.h"

__global__ void bitonicSortShared1(
    uint *d_DstKey,
    uint *d_DstVal,
    uint *d_SrcKey,
    uint *d_SrcVal
)
{
    //Shared memory storage for current subarray
    __shared__ uint s_key[SHARED_SIZE_LIMIT];
    __shared__ uint s_val[SHARED_SIZE_LIMIT];

    //Offset to the beginning of subarray and load data
    d_SrcKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_SrcVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    s_key[threadIdx.x +                       0] = d_SrcKey[                      0];
    s_val[threadIdx.x +                       0] = d_SrcVal[                      0];
    s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcKey[(SHARED_SIZE_LIMIT / 2)];
    s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcVal[(SHARED_SIZE_LIMIT / 2)];

    for (uint size = 2; size < SHARED_SIZE_LIMIT; size <<= 1)
    {
        //Bitonic merge
        uint ddd = (threadIdx.x & (size / 2)) != 0;

        for (uint stride = size / 2; stride > 0; stride >>= 1)
        {
            __syncthreads();
            uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            Comparator(
                s_key[pos +      0], s_val[pos +      0],
                s_key[pos + stride], s_val[pos + stride],
                ddd
            );
        }
    }

    //Odd / even arrays of SHARED_SIZE_LIMIT elements
    //sorted in opposite directions
    uint ddd = blockIdx.x & 1;
    {
        for (uint stride = SHARED_SIZE_LIMIT / 2; stride > 0; stride >>= 1)
        {
            __syncthreads();
            uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            Comparator(
                s_key[pos +      0], s_val[pos +      0],
                s_key[pos + stride], s_val[pos + stride],
                ddd
            );
        }
    }


    __syncthreads();
    d_DstKey[                      0] = s_key[threadIdx.x +                       0];
    d_DstVal[                      0] = s_val[threadIdx.x +                       0];
    d_DstKey[(SHARED_SIZE_LIMIT / 2)] = s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
    d_DstVal[(SHARED_SIZE_LIMIT / 2)] = s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
}
