#include "hip/hip_runtime.h"
//pass
//--gridDim=14 --blockDim=[32,32] --only-intra-group

template<class T>
__global__ void Mandelbrot1(uchar4 *dst, const int imageW, const int imageH, const int crunch, const T xOff, const T yOff,
                            const T xJP, const T yJP, const T scale, const uchar4 colors, const int frame,
                            const int animationFrame, const int gridWidth, const int numBlocks, const bool isJ);
template
__global__ void Mandelbrot1<float>(uchar4 *dst, const int imageW, const int imageH, const int crunch, const float xOff, const float yOff,
                            const float xJP, const float yJP, const float scale, const uchar4 colors, const int frame,
                            const int animationFrame, const int gridWidth, const int numBlocks, const bool isJ);

#include "common.h"

// The Mandelbrot secondary AA pass CUDA GPU thread function
template<class T>
__global__ void Mandelbrot1(uchar4 *dst, const int imageW, const int imageH, const int crunch, const T xOff, const T yOff,
                            const T xJP, const T yJP, const T scale, const uchar4 colors, const int frame,
                            const int animationFrame, const int gridWidth, const int numBlocks, const bool isJ)
{
    __requires(imageW == 800);
    __requires(imageH == 600);
    __requires(gridWidth == 25);
    __requires(numBlocks == 475);

    __shared__ unsigned int blockIndex;
    __shared__ unsigned int blockX, blockY;

    // loop until all blocks completed
    while (1)
    {
#ifndef KERNEL_BUG
        __syncthreads();
#endif

        if ((threadIdx.x==0) && (threadIdx.y==0))
        {
            // get block to process
            blockIndex = atomicAdd(&blockCounter, 1);
            blockX = blockIndex % gridWidth;            // note: this is slow, but only called once per block here
            blockY = blockIndex / gridWidth;
        }

        __syncthreads();

        if (blockIndex >= numBlocks)
        {
            break;    // finish
        }

        // process this block
        const int ix = blockDim.x * blockX + threadIdx.x;
        const int iy = blockDim.y * blockY + threadIdx.y;

        if ((ix < imageW) && (iy < imageH))
        {
            // Get the current pixel color
            int pixel = imageW * iy + ix;
            uchar4 pixelColor = dst[pixel];
            int count = 0;

            // Search for pixels out of tolerance surrounding the current pixel
            if (ix > 0)
            {
                count += CheckColors(pixelColor, dst[pixel - 1]);
            }

            if (ix + 1 < imageW)
            {
                count += CheckColors(pixelColor, dst[pixel + 1]);
            }

            if (iy > 0)
            {
                count += CheckColors(pixelColor, dst[pixel - imageW]);
            }

            if (iy + 1 < imageH)
            {
                count += CheckColors(pixelColor, dst[pixel + imageW]);
            }

            if (count)
            {
                // Calculate the location
                const T xPos = (T)ix * scale + xOff;
                const T yPos = (T)iy * scale + yOff;

                // Calculate the Mandelbrot index for the current location
                int m = CalcMandelbrot(xPos, yPos, xJP, yJP, crunch, isJ);
                m = m > 0 ? crunch - m : 0;

                // Convert the Mandelbrot index into a color
                uchar4 color;

                if (m)
                {
                    m += animationFrame;
                    color.x = m * colors.x;
                    color.y = m * colors.y;
                    color.z = m * colors.z;
                }
                else
                {
                    color.x = 0;
                    color.y = 0;
                    color.z = 0;
                }

                // Output the pixel
                int frame1 = frame + 1;
                int frame2 = frame1 / 2;
                dst[pixel].x = (pixelColor.x * frame + color.x + frame2) / frame1;
                dst[pixel].y = (pixelColor.y * frame + color.y + frame2) / frame1;
                dst[pixel].z = (pixelColor.z * frame + color.z + frame2) / frame1;
            }
        }
    }

} // Mandelbrot1
