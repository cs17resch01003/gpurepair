#include "hip/hip_runtime.h"
//pass
//--gridDim=26               --blockDim=256

#include "common.h"

__global__ void scanExclusiveShared2(
    uint *d_Buf,
    uint *d_Dst,
    uint *d_Src,
    uint N,
    uint arrayLength
)
{
    __requires(N == 6656);
    __requires((arrayLength & (arrayLength - 1)) == 0);
    __shared__ uint s_Data[2 * THREADBLOCK_SIZE];

    //Skip loads and stores for inactive threads of last threadblock (pos >= N)
    uint pos = blockIdx.x * blockDim.x + threadIdx.x;

    //Load top elements
    //Convert results of bottom-level scan back to inclusive
    uint idata = 0;

    if (pos < N)
        idata =
            d_Dst[(4 * THREADBLOCK_SIZE) - 1 + (4 * THREADBLOCK_SIZE) * pos] +
            d_Src[(4 * THREADBLOCK_SIZE) - 1 + (4 * THREADBLOCK_SIZE) * pos];

    //Compute
    uint odata = scan1Exclusive(idata, s_Data, arrayLength);

    //Avoid out-of-bound access
    if (pos < N)
    {
        d_Buf[pos] = odata;
    }
}
