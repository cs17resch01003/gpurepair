#include "hip/hip_runtime.h"
//pass
//--gridDim=32768            --blockDim=128

#include "common_merge.h"
template<uint sortDir> __global__ void mergeElementaryIntervalsKernel(uint *d_DstKey, uint *d_DstVal, uint *d_SrcKey, uint *d_SrcVal, uint *d_LimitsA, uint *d_LimitsB, uint stride, uint N);
template               __global__ void mergeElementaryIntervalsKernel<1>(uint *d_DstKey, uint *d_DstVal, uint *d_SrcKey, uint *d_SrcVal, uint *d_LimitsA, uint *d_LimitsB, uint stride, uint N);

template<uint sortDir> __device__ static __attribute__((always_inline)) void merge(
    uint *dstKey,
    uint *dstVal,
    uint *srcAKey,
    uint *srcAVal,
    uint *srcBKey,
    uint *srcBVal,
    uint lenA,
    uint nPowTwoLenA,
    uint lenB,
    uint nPowTwoLenB
)
{
    uint keyA, valA, keyB, valB, dstPosA, dstPosB;

    if (threadIdx.x < lenA)
    {
        keyA = srcAKey[threadIdx.x];
        valA = srcAVal[threadIdx.x];
        dstPosA = binarySearchExclusive<sortDir>(keyA, srcBKey, lenB, nPowTwoLenB) + threadIdx.x;
    }

    if (threadIdx.x < lenB)
    {
        keyB = srcBKey[threadIdx.x];
        valB = srcBVal[threadIdx.x];
        dstPosB = binarySearchInclusive<sortDir>(keyB, srcAKey, lenA, nPowTwoLenA) + threadIdx.x;
    }

    __syncthreads();

    if (threadIdx.x < lenA)
    {
        dstKey[dstPosA] = keyA;
        dstVal[dstPosA] = valA;
    }

    if (threadIdx.x < lenB)
    {
        dstKey[dstPosB] = keyB;
        dstVal[dstPosB] = valB;
    }
}


template<uint sortDir> __global__ void mergeElementaryIntervalsKernel(
    uint *d_DstKey,
    uint *d_DstVal,
    uint *d_SrcKey,
    uint *d_SrcVal,
    uint *d_LimitsA,
    uint *d_LimitsB,
    uint stride,
    uint N
)
{
    __requires((stride & (stride - 1)) == 0);
    __requires(stride < N);
    __requires(stride > 1);
    __shared__ uint s_key[2 * SAMPLE_STRIDE];
    __shared__ uint s_val[2 * SAMPLE_STRIDE];

    const uint   intervalI = blockIdx.x & ((2 * stride) / SAMPLE_STRIDE - 1);
    const uint segmentBase = (blockIdx.x - intervalI) * SAMPLE_STRIDE;
    d_SrcKey += segmentBase;
    d_SrcVal += segmentBase;
    d_DstKey += segmentBase;
    d_DstVal += segmentBase;

    //Set up threadblock-wide parameters
    __shared__ uint startSrcA, startSrcB, lenSrcA, lenSrcB, startDstA, startDstB;

    if (threadIdx.x == 0)
    {
        uint segmentElementsA = stride;
        uint segmentElementsB = umin(stride, N - segmentBase - stride);
        uint  segmentSamplesA = getSampleCount(segmentElementsA);
        uint  segmentSamplesB = getSampleCount(segmentElementsB);
        uint   segmentSamples = segmentSamplesA + segmentSamplesB;

        startSrcA    = d_LimitsA[blockIdx.x];
        startSrcB    = d_LimitsB[blockIdx.x];
        uint endSrcA = (intervalI + 1 < segmentSamples) ? d_LimitsA[blockIdx.x + 1] : segmentElementsA;
        uint endSrcB = (intervalI + 1 < segmentSamples) ? d_LimitsB[blockIdx.x + 1] : segmentElementsB;
        lenSrcA      = endSrcA - startSrcA;
        lenSrcB      = endSrcB - startSrcB;
        startDstA    = startSrcA + startSrcB;
        startDstB    = startDstA + lenSrcA;
    }

    //Load main input data
    __syncthreads();

    if (threadIdx.x < lenSrcA)
    {
        s_key[threadIdx.x +             0] = d_SrcKey[0 + startSrcA + threadIdx.x];
        s_val[threadIdx.x +             0] = d_SrcVal[0 + startSrcA + threadIdx.x];
    }

    if (threadIdx.x < lenSrcB)
    {
        s_key[threadIdx.x + SAMPLE_STRIDE] = d_SrcKey[stride + startSrcB + threadIdx.x];
        s_val[threadIdx.x + SAMPLE_STRIDE] = d_SrcVal[stride + startSrcB + threadIdx.x];
    }

    //Merge data in shared memory
    __syncthreads();
    merge<sortDir>(
        s_key,
        s_val,
        s_key + 0,
        s_val + 0,
        s_key + SAMPLE_STRIDE,
        s_val + SAMPLE_STRIDE,
        lenSrcA, SAMPLE_STRIDE,
        lenSrcB, SAMPLE_STRIDE
    );

    //Store merged data
    __syncthreads();

    if (threadIdx.x < lenSrcA)
    {
        d_DstKey[startDstA + threadIdx.x] = s_key[threadIdx.x];
        d_DstVal[startDstA + threadIdx.x] = s_val[threadIdx.x];
    }

    if (threadIdx.x < lenSrcB)
    {
        d_DstKey[startDstB + threadIdx.x] = s_key[lenSrcA + threadIdx.x];
        d_DstVal[startDstB + threadIdx.x] = s_val[lenSrcA + threadIdx.x];
    }
}
