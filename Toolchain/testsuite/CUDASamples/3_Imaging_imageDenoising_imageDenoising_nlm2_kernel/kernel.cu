#include "hip/hip_runtime.h"
//xfail:NOT_ALL_VERIFIED
//--gridDim=[40,51] --blockDim=[8,8]

#include "common.h"

__global__ void NLM2(
    TColor *dst,
    int imageW,
    int imageH,
    float Noise,
    float lerpC
)
{
    __requires(imageW == 320);
    __requires(imageH == 408);
    //Weights cache
    __shared__ float fWeights[BLOCKDIM_X * BLOCKDIM_Y];

    const int   ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int   iy = blockDim.y * blockIdx.y + threadIdx.y;
    //Add half of a texel to always address exact texel centers
    const float  x = (float)ix  + 0.5f;
    const float  y = (float)iy  + 0.5f;
    const float cx = blockDim.x * blockIdx.x + NLM_WINDOW_RADIUS + 0.5f;
    const float cy = blockDim.x * blockIdx.y + NLM_WINDOW_RADIUS + 0.5f;

    if (ix < imageW && iy < imageH)
    {
        //Find color distance from current texel to the center of NLM window
        float weight = 0;

        for (float n = -NLM_BLOCK_RADIUS; n <= NLM_BLOCK_RADIUS; n++){
            for (float m = -NLM_BLOCK_RADIUS; m <= NLM_BLOCK_RADIUS; m++){
                weight += vecLen(
                              tex2D(texImage, cx + m, cy + n),
                              tex2D(texImage,  x + m,  y + n)
                          );
            }
        }

        //Geometric distance from current texel to the center of NLM window
        float dist =
            (threadIdx.x - NLM_WINDOW_RADIUS) * (threadIdx.x - NLM_WINDOW_RADIUS) +
            (threadIdx.y - NLM_WINDOW_RADIUS) * (threadIdx.y - NLM_WINDOW_RADIUS);

        //Derive final weight from color and geometric distance
        weight = __expf(-(weight * Noise + dist * INV_NLM_WINDOW_AREA));

        //Write the result to shared memory
        fWeights[threadIdx.y * BLOCKDIM_X + threadIdx.x] = weight;
        //Wait until all the weights are ready
        // __syncthreads();


        //Normalized counter for the NLM weight threshold
        float fCount = 0;
        //Total sum of pixel weights
        float sumWeights = 0;
        //Result accumulator
        float3 clr = {0, 0, 0};

        int idx = 0;

        //Cycle through NLM window, surrounding (x, y) texel
        for (float i = -NLM_WINDOW_RADIUS; i <= NLM_WINDOW_RADIUS + 1; i++)
            for (float j = -NLM_WINDOW_RADIUS; j <= NLM_WINDOW_RADIUS + 1; j++)
            {
                //Load precomputed weight
                float weightIJ = fWeights[idx++];

                //Accumulate (x + j, y + i) texel color with computed weight
                float4 clrIJ = tex2D(texImage, x + j, y + i);
                clr.x       += clrIJ.x * weightIJ;
                clr.y       += clrIJ.y * weightIJ;
                clr.z       += clrIJ.z * weightIJ;

                //Sum of weights for color normalization to [0..1] range
                sumWeights  += weightIJ;

                //Update weight counter, if NLM weight for current window texel
                //exceeds the weight threshold
                fCount      += (weightIJ > NLM_WEIGHT_THRESHOLD) ? INV_NLM_WINDOW_AREA : 0;
            }

        //Normalize result color by sum of weights
        sumWeights = 1.0f / sumWeights;
        clr.x *= sumWeights;
        clr.y *= sumWeights;
        clr.z *= sumWeights;

        //Choose LERP quotent basing on how many texels
        //within the NLM window exceeded the weight threshold
        float lerpQ = (fCount > NLM_LERP_THRESHOLD) ? lerpC : 1.0f - lerpC;

        //Write final result to global memory
        float4 clr00 = tex2D(texImage, x, y);
        clr.x = lerpf(clr.x, clr00.x, lerpQ);
        clr.y = lerpf(clr.y, clr00.y, lerpQ);
        clr.z = lerpf(clr.z, clr00.z, lerpQ);
        dst[imageW * iy + ix] = make_color(clr.x, clr.y, clr.z, 0);
    }
}
