#include "hip/hip_runtime.h"
//pass
//--gridDim=8 --blockDim=512

#define SHARED_SIZE_LIMIT 1024U
#define     SAMPLE_STRIDE 128
typedef unsigned int uint;

__device__ static __attribute__((always_inline)) void Comparator(
    uint &keyA,
    uint &valA,
    uint &keyB,
    uint &valB,
    uint arrowDir
)
{
    uint t;

    if ((keyA > keyB) == arrowDir)
    {
        t = keyA;
        keyA = keyB;
        keyB = t;
        t = valA;
        valA = valB;
        valB = t;
    }
}

__global__ void bitonicSortSharedKernel(
    uint *d_DstKey,
    uint *d_DstVal,
    uint *d_SrcKey,
    uint *d_SrcVal,
    uint arrayLength,
    uint sortDir
)
{
    __requires(arrayLength & (arrayLength - 1) == 0);
    //Shared memory storage for one or more short vectors
    __shared__ uint s_key[SHARED_SIZE_LIMIT];
    __shared__ uint s_val[SHARED_SIZE_LIMIT];

    //Offset to the beginning of subbatch and load data
    d_SrcKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_SrcVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    s_key[threadIdx.x +                       0] = d_SrcKey[                      0];
    s_val[threadIdx.x +                       0] = d_SrcVal[                      0];
    s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcKey[(SHARED_SIZE_LIMIT / 2)];
    s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcVal[(SHARED_SIZE_LIMIT / 2)];

    for (uint size = 2; size < arrayLength; size <<= 1)
    {
        //Bitonic merge
        uint dir = (threadIdx.x & (size / 2)) != 0;

        for (uint stride = size / 2; stride > 0; stride >>= 1)
        {
            __syncthreads();
            uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            Comparator(
                s_key[pos +      0], s_val[pos +      0],
                s_key[pos + stride], s_val[pos + stride],
                dir
            );
        }
    }

    //ddd == sortDir for the last bitonic merge step
    {
        for (uint stride = arrayLength / 2; stride > 0; stride >>= 1)
        {
            __syncthreads();
            uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            Comparator(
                s_key[pos +      0], s_val[pos +      0],
                s_key[pos + stride], s_val[pos + stride],
                sortDir
            );
        }
    }

    __syncthreads();
    d_DstKey[                      0] = s_key[threadIdx.x +                       0];
    d_DstVal[                      0] = s_val[threadIdx.x +                       0];
    d_DstKey[(SHARED_SIZE_LIMIT / 2)] = s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
    d_DstVal[(SHARED_SIZE_LIMIT / 2)] = s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
}
