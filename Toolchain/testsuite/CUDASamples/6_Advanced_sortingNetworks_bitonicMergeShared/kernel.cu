#include "hip/hip_runtime.h"
//pass
//--gridDim=1024             --blockDim=512 2

#include "common.h"

__global__ void bitonicMergeShared(
    uint *d_DstKey,
    uint *d_DstVal,
    uint *d_SrcKey,
    uint *d_SrcVal,
    uint arrayLength,
    uint size,
    uint dir
)
{
    __requires(arrayLength == 2048);
    __requires(size == 1024);
    
    //Shared memory storage for current subarray
    __shared__ uint s_key[SHARED_SIZE_LIMIT];
    __shared__ uint s_val[SHARED_SIZE_LIMIT];

    d_SrcKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_SrcVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    s_key[threadIdx.x +                       0] = d_SrcKey[                      0];
    s_val[threadIdx.x +                       0] = d_SrcVal[                      0];
    s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcKey[(SHARED_SIZE_LIMIT / 2)];
    s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcVal[(SHARED_SIZE_LIMIT / 2)];

    //Bitonic merge
    uint comparatorI = UMAD(blockIdx.x, blockDim.x, threadIdx.x) & ((arrayLength / 2) - 1);
    uint ddd = dir ^ ((comparatorI & (size / 2)) != 0);

    for (uint stride = SHARED_SIZE_LIMIT / 2; stride > 0; stride >>= 1)
    {
        // __syncthreads();
        uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
        Comparator(
            s_key[pos +      0], s_val[pos +      0],
            s_key[pos + stride], s_val[pos + stride],
            ddd
        );
    }

    // __syncthreads();
    d_DstKey[                      0] = s_key[threadIdx.x +                       0];
    d_DstVal[                      0] = s_val[threadIdx.x +                       0];
    d_DstKey[(SHARED_SIZE_LIMIT / 2)] = s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
    d_DstVal[(SHARED_SIZE_LIMIT / 2)] = s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
}
