#include "hip/hip_runtime.h"
//pass
//--gridDim=4096             --blockDim=512

#include "common_merge.h"

template<uint sortDir> __global__ void mergeSortSharedKernel(uint *d_DstKey, uint *d_DstVal, uint *d_SrcKey, uint *d_SrcVal, uint arrayLength);
template               __global__ void mergeSortSharedKernel<1>(uint *d_DstKey, uint *d_DstVal, uint *d_SrcKey, uint *d_SrcVal, uint arrayLength);

template<uint sortDir> __global__ void mergeSortSharedKernel(
    uint *d_DstKey,
    uint *d_DstVal,
    uint *d_SrcKey,
    uint *d_SrcVal,
    uint arrayLength
)
{
    __shared__ uint s_key[SHARED_SIZE_LIMIT];
    __shared__ uint s_val[SHARED_SIZE_LIMIT];

    d_SrcKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_SrcVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    s_key[threadIdx.x +                       0] = d_SrcKey[                      0];
    s_val[threadIdx.x +                       0] = d_SrcVal[                      0];
    s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcKey[(SHARED_SIZE_LIMIT / 2)];
    s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcVal[(SHARED_SIZE_LIMIT / 2)];

    for (uint stride = 1; stride < arrayLength; stride <<= 1)
    {
        uint     lPos = threadIdx.x & (stride - 1);
        uint *baseKey = s_key + 2 * (threadIdx.x - lPos);
        uint *baseVal = s_val + 2 * (threadIdx.x - lPos);

        __syncthreads();
        uint keyA = baseKey[lPos +      0];
        uint valA = baseVal[lPos +      0];
        uint keyB = baseKey[lPos + stride];
        uint valB = baseVal[lPos + stride];
        uint posA = binarySearchExclusive<sortDir>(keyA, baseKey + stride, stride, stride) + lPos;
        uint posB = binarySearchInclusive<sortDir>(keyB, baseKey +      0, stride, stride) + lPos;

        __syncthreads();
        baseKey[posA] = keyA;
        baseVal[posA] = valA;
        baseKey[posB] = keyB;
        baseVal[posB] = valB;
    }

    __syncthreads();
    d_DstKey[                      0] = s_key[threadIdx.x +                       0];
    d_DstVal[                      0] = s_val[threadIdx.x +                       0];
    d_DstKey[(SHARED_SIZE_LIMIT / 2)] = s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
    d_DstVal[(SHARED_SIZE_LIMIT / 2)] = s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
}
