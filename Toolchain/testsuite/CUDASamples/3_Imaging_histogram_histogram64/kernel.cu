#include "hip/hip_runtime.h"
//pass
//--gridDim=4370             --blockDim=64

#include "common.h"

//Data type used for input data fetches
typedef uint4 data_t;

//Count a byte into shared-memory storage
__device__ static __attribute__((always_inline)) void addByte(uchar *s_ThreadBase, uint data)
{
    s_ThreadBase[UMUL(data, HISTOGRAM64_THREADBLOCK_SIZE)]++;
}

//Count four bytes of a word
__device__ static __attribute__((always_inline)) void addWord(uchar *s_ThreadBase, uint data)
{
    //Only higher 6 bits of each byte matter, as this is a 64-bin histogram
    addByte(s_ThreadBase, (data >>  2) & 0x3FU);
    addByte(s_ThreadBase, (data >> 10) & 0x3FU);
    addByte(s_ThreadBase, (data >> 18) & 0x3FU);
    addByte(s_ThreadBase, (data >> 26) & 0x3FU);
}

__global__ void histogram64Kernel(uint *d_PartialHistograms, data_t *d_Data, uint dataCount)
{
    //Encode thread index in order to avoid bank conflicts in s_Hist[] access:
    //each group of SHARED_MEMORY_BANKS threads accesses consecutive shared memory banks
    //and the same bytes [0..3] within the banks
    //Because of this permutation block size should be a multiple of 4 * SHARED_MEMORY_BANKS
    const uint threadPos =
        ((threadIdx.x & ~(SHARED_MEMORY_BANKS * 4 - 1)) << 0) |
        ((threadIdx.x & (SHARED_MEMORY_BANKS     - 1)) << 2) |
        ((threadIdx.x & (SHARED_MEMORY_BANKS * 3)) >> 4);

    //Per-thread histogram storage
    __shared__ uchar s_Hist[HISTOGRAM64_THREADBLOCK_SIZE * HISTOGRAM64_BIN_COUNT];
    uchar *s_ThreadBase = s_Hist + threadPos;

    //Initialize shared memory (writing 32-bit words)
#pragma unroll

    for (uint i = 0; i < (HISTOGRAM64_BIN_COUNT / 4); i++)
    {
        ((uint *)s_Hist)[threadIdx.x + i * HISTOGRAM64_THREADBLOCK_SIZE] = 0;
    }

    //Read data from global memory and submit to the shared-memory histogram
    //Since histogram counters are byte-sized, every single thread can't do more than 255 submission
    // __syncthreads();

    for (uint pos = UMAD(blockIdx.x, blockDim.x, threadIdx.x);__global_invariant(__write_implies(s_Hist, (__write_offset_bytes(s_Hist)/sizeof(uchar) - threadPos)%HISTOGRAM64_THREADBLOCK_SIZE == 0)),__global_invariant(__read_implies(s_Hist, (__read_offset_bytes(s_Hist)/sizeof(uchar) - threadPos)%HISTOGRAM64_THREADBLOCK_SIZE == 0)),         pos < dataCount; pos += UMUL(blockDim.x, gridDim.x))
    {
        data_t data = d_Data[pos];
        addWord(s_ThreadBase, data.x);
        addWord(s_ThreadBase, data.y);
        addWord(s_ThreadBase, data.z);
        addWord(s_ThreadBase, data.w);
    }

    //Accumulate per-thread histograms into per-block and write to global memory
    // __syncthreads();

    if (threadIdx.x < HISTOGRAM64_BIN_COUNT)
    {
        uchar *s_HistBase = s_Hist + UMUL(threadIdx.x, HISTOGRAM64_THREADBLOCK_SIZE);

        uint sum = 0;
        uint pos = 4 * (threadIdx.x & (SHARED_MEMORY_BANKS - 1));

#pragma unroll

        for (uint i = 0; i < (HISTOGRAM64_THREADBLOCK_SIZE / 4); i++)
        {
            sum +=
                s_HistBase[pos + 0] +
                s_HistBase[pos + 1] +
                s_HistBase[pos + 2] +
                s_HistBase[pos + 3];
            pos = (pos + 4) & (HISTOGRAM64_THREADBLOCK_SIZE - 1);
        }

        d_PartialHistograms[blockIdx.x * HISTOGRAM64_BIN_COUNT + threadIdx.x] = sum;
    }
}
