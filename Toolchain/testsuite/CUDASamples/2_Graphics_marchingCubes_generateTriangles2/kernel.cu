#include "hip/hip_runtime.h"
//pass
//--gridDim=[52,1,1] --blockDim=32

#include "common.h"

__global__ void
generateTriangles2(float4 *pos, float4 *norm, uint *compactedVoxelArray, uint *numVertsScanned, uchar *volume,
                   uint3 gridSize, uint3 gridSizeShift, uint3 gridSizeMask,
                   float3 voxelSize, float isoValue, uint activeVoxels, uint maxVerts)
{
    uint blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
    uint i = __mul24(blockId, blockDim.x) + threadIdx.x;

    if (i > activeVoxels - 1)
    {
        i = activeVoxels - 1;
    }

#if SKIP_EMPTY_VOXELS
    uint voxel = compactedVoxelArray[i];
#else
    uint voxel = i;
#endif

    // compute position in 3d grid
    uint3 gridPos = calcGridPos(voxel, gridSizeShift, gridSizeMask);

    float3 p;
    p.x = -1.0f + (gridPos.x * voxelSize.x);
    p.y = -1.0f + (gridPos.y * voxelSize.y);
    p.z = -1.0f + (gridPos.z * voxelSize.z);

    // calculate cell vertex positions
    float3 v[8];
    v[0] = p;
    v[1] = p + make_float3(voxelSize.x, 0, 0);
    v[2] = p + make_float3(voxelSize.x, voxelSize.y, 0);
    v[3] = p + make_float3(0, voxelSize.y, 0);
    v[4] = p + make_float3(0, 0, voxelSize.z);
    v[5] = p + make_float3(voxelSize.x, 0, voxelSize.z);
    v[6] = p + make_float3(voxelSize.x, voxelSize.y, voxelSize.z);
    v[7] = p + make_float3(0, voxelSize.y, voxelSize.z);

#if SAMPLE_VOLUME
    float field[8];
    field[0] = sampleVolume(volume, gridPos, gridSize);
    field[1] = sampleVolume(volume, gridPos + make_uint3(1, 0, 0), gridSize);
    field[2] = sampleVolume(volume, gridPos + make_uint3(1, 1, 0), gridSize);
    field[3] = sampleVolume(volume, gridPos + make_uint3(0, 1, 0), gridSize);
    field[4] = sampleVolume(volume, gridPos + make_uint3(0, 0, 1), gridSize);
    field[5] = sampleVolume(volume, gridPos + make_uint3(1, 0, 1), gridSize);
    field[6] = sampleVolume(volume, gridPos + make_uint3(1, 1, 1), gridSize);
    field[7] = sampleVolume(volume, gridPos + make_uint3(0, 1, 1), gridSize);
#else
    // evaluate field values
    float field[8];
    field[0] = fieldFunc(v[0]);
    field[1] = fieldFunc(v[1]);
    field[2] = fieldFunc(v[2]);
    field[3] = fieldFunc(v[3]);
    field[4] = fieldFunc(v[4]);
    field[5] = fieldFunc(v[5]);
    field[6] = fieldFunc(v[6]);
    field[7] = fieldFunc(v[7]);
#endif

    // recalculate flag
    uint cubeindex;
    cubeindex =  uint(field[0] < isoValue);
    cubeindex += uint(field[1] < isoValue)*2;
    cubeindex += uint(field[2] < isoValue)*4;
    cubeindex += uint(field[3] < isoValue)*8;
    cubeindex += uint(field[4] < isoValue)*16;
    cubeindex += uint(field[5] < isoValue)*32;
    cubeindex += uint(field[6] < isoValue)*64;
    cubeindex += uint(field[7] < isoValue)*128;

    // find the vertices where the surface intersects the cube

#if USE_SHARED
    // use shared memory to avoid using local
    __shared__ float3 vertlist[12*NTHREADS];

    vertlist[threadIdx.x] = vertexInterp(isoValue, v[0], v[1], field[0], field[1]);
    vertlist[NTHREADS+threadIdx.x] = vertexInterp(isoValue, v[1], v[2], field[1], field[2]);
    vertlist[(NTHREADS*2)+threadIdx.x] = vertexInterp(isoValue, v[2], v[3], field[2], field[3]);
    vertlist[(NTHREADS*3)+threadIdx.x] = vertexInterp(isoValue, v[3], v[0], field[3], field[0]);
    vertlist[(NTHREADS*4)+threadIdx.x] = vertexInterp(isoValue, v[4], v[5], field[4], field[5]);
    vertlist[(NTHREADS*5)+threadIdx.x] = vertexInterp(isoValue, v[5], v[6], field[5], field[6]);
    vertlist[(NTHREADS*6)+threadIdx.x] = vertexInterp(isoValue, v[6], v[7], field[6], field[7]);
    vertlist[(NTHREADS*7)+threadIdx.x] = vertexInterp(isoValue, v[7], v[4], field[7], field[4]);
    vertlist[(NTHREADS*8)+threadIdx.x] = vertexInterp(isoValue, v[0], v[4], field[0], field[4]);
    vertlist[(NTHREADS*9)+threadIdx.x] = vertexInterp(isoValue, v[1], v[5], field[1], field[5]);
    vertlist[(NTHREADS*10)+threadIdx.x] = vertexInterp(isoValue, v[2], v[6], field[2], field[6]);
    vertlist[(NTHREADS*11)+threadIdx.x] = vertexInterp(isoValue, v[3], v[7], field[3], field[7]);
    // __syncthreads();
#else

    float3 vertlist[12];

    vertlist[0] = vertexInterp(isoValue, v[0], v[1], field[0], field[1]);
    vertlist[1] = vertexInterp(isoValue, v[1], v[2], field[1], field[2]);
    vertlist[2] = vertexInterp(isoValue, v[2], v[3], field[2], field[3]);
    vertlist[3] = vertexInterp(isoValue, v[3], v[0], field[3], field[0]);

    vertlist[4] = vertexInterp(isoValue, v[4], v[5], field[4], field[5]);
    vertlist[5] = vertexInterp(isoValue, v[5], v[6], field[5], field[6]);
    vertlist[6] = vertexInterp(isoValue, v[6], v[7], field[6], field[7]);
    vertlist[7] = vertexInterp(isoValue, v[7], v[4], field[7], field[4]);

    vertlist[8] = vertexInterp(isoValue, v[0], v[4], field[0], field[4]);
    vertlist[9] = vertexInterp(isoValue, v[1], v[5], field[1], field[5]);
    vertlist[10] = vertexInterp(isoValue, v[2], v[6], field[2], field[6]);
    vertlist[11] = vertexInterp(isoValue, v[3], v[7], field[3], field[7]);
#endif

    // output triangle vertices
    uint numVerts = tex1Dfetch(numVertsTex, cubeindex);

    for (int i=0; i<numVerts; i+=3)
    {
        uint index = numVertsScanned[voxel] + i;

        // imperial edit: replaced array of three float3 pointers with three distinct float3 pointers
        
        float3* v0;
        float3* v1;
        float3* v2;

        uint edge;
        edge = tex1Dfetch(triTex, (cubeindex*16) + i);
#if USE_SHARED
        v0 = &vertlist[(edge*NTHREADS)+threadIdx.x];
#else
        v0 = &vertlist[edge];
#endif

        edge = tex1Dfetch(triTex, (cubeindex*16) + i + 1);
#if USE_SHARED
        v1 = &vertlist[(edge*NTHREADS)+threadIdx.x];
#else
        v1 = &vertlist[edge];
#endif

        edge = tex1Dfetch(triTex, (cubeindex*16) + i + 2);
#if USE_SHARED
        v2 = &vertlist[(edge*NTHREADS)+threadIdx.x];
#else
        v2 = &vertlist[edge];
#endif

        // calculate triangle surface normal
        float3 n = calcNormal(v0, v1, v2);

        if (index < (maxVerts - 3))
        {
#if 0
            pos[index] = make_float4(*v[0], 1.0f);
            norm[index] = make_float4(n, 0.0f);

            pos[index+1] = make_float4(*v[1], 1.0f);
            norm[index+1] = make_float4(n, 0.0f);

            pos[index+2] = make_float4(*v[2], 1.0f);
            norm[index+2] = make_float4(n, 0.0f);
#else
            float4 v1, v2, v3, v4, v5, v6;
            pos[index] = v1;
            norm[index] = v2;

            pos[index+1] = v3;
            norm[index+1] = v4;

            pos[index+2] = v5;
            norm[index+2] = v6;
#endif
        }
    }
}
