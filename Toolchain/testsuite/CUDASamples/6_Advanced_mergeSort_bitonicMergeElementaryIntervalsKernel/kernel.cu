#include "hip/hip_runtime.h"
//pass
//--gridDim=8 --blockDim=128

#define umin(x,y) (x < y ? x : y)
#define     SAMPLE_STRIDE 128
typedef unsigned int uint;
template<uint sortDir> __global__ void bitonicMergeElementaryIntervalsKernel(uint *d_DstKey, uint *d_DstVal, uint *d_SrcKey, uint *d_SrcVal, uint *d_LimitsA, uint *d_LimitsB, uint stride, uint N);
template               __global__ void bitonicMergeElementaryIntervalsKernel<1>(uint *d_DstKey, uint *d_DstVal, uint *d_SrcKey, uint *d_SrcVal, uint *d_LimitsA, uint *d_LimitsB, uint stride, uint N);

__device__ static __attribute__((always_inline)) uint iDivUp(uint a, uint b)
{
    return ((a % b) == 0) ? (a / b) : (a / b + 1);
}

__device__ static __attribute__((always_inline)) uint getSampleCount(uint dividend)
{
    return iDivUp(dividend, SAMPLE_STRIDE);
}

template<uint sortDir> __device__ static __attribute__((always_inline)) void ComparatorExtended(
    uint &keyA,
    uint &valA,
    uint &flagA,
    uint &keyB,
    uint &valB,
    uint &flagB,
    uint arrowDir
)
{
    uint t;

    if (
        (!(flagA || flagB) && ((keyA > keyB) == arrowDir)) ||
        ((arrowDir == sortDir) && (flagA == 1)) ||
        ((arrowDir != sortDir) && (flagB == 1))
    )
    {
        t = keyA;
        keyA = keyB;
        keyB = t;
        t = valA;
        valA = valB;
        valB = t;
        t = flagA;
        flagA = flagB;
        flagB = t;
    }
}

template<uint sortDir> __global__ void bitonicMergeElementaryIntervalsKernel(
    uint *d_DstKey,
    uint *d_DstVal,
    uint *d_SrcKey,
    uint *d_SrcVal,
    uint *d_LimitsA,
    uint *d_LimitsB,
    uint stride,
    uint N
)
{
    __requires((stride & (stride - 1)) == 0);
    __requires(stride < N);
    __requires(stride > 1);
    __shared__ uint s_key[2 * SAMPLE_STRIDE];
    __shared__ uint s_val[2 * SAMPLE_STRIDE];
    __shared__ uint s_inf[2 * SAMPLE_STRIDE];

    const uint   intervalI = blockIdx.x & ((2 * stride) / SAMPLE_STRIDE - 1);
    const uint segmentBase = (blockIdx.x - intervalI) * SAMPLE_STRIDE;
    d_SrcKey += segmentBase;
    d_SrcVal += segmentBase;
    d_DstKey += segmentBase;
    d_DstVal += segmentBase;

    //Set up threadblock-wide parameters
    __shared__ uint startSrcA, lenSrcA, startSrcB, lenSrcB, startDst;

    if (threadIdx.x == 0)
    {
        uint segmentElementsA = stride;
        uint segmentElementsB = umin(stride, N - segmentBase - stride);
        uint  segmentSamplesA = stride / SAMPLE_STRIDE;
        uint  segmentSamplesB = getSampleCount(segmentElementsB);
        uint   segmentSamples = segmentSamplesA + segmentSamplesB;

        startSrcA    = d_LimitsA[blockIdx.x];
        startSrcB    = d_LimitsB[blockIdx.x];
        startDst     = startSrcA + startSrcB;

        uint endSrcA = (intervalI + 1 < segmentSamples) ? d_LimitsA[blockIdx.x + 1] : segmentElementsA;
        uint endSrcB = (intervalI + 1 < segmentSamples) ? d_LimitsB[blockIdx.x + 1] : segmentElementsB;
        lenSrcA      = endSrcA - startSrcA;
        lenSrcB      = endSrcB - startSrcB;
    }

    s_inf[threadIdx.x +             0] = 1;
    s_inf[threadIdx.x + SAMPLE_STRIDE] = 1;

    //Load input data
    __syncthreads();

    if (threadIdx.x < lenSrcA)
    {
        s_key[threadIdx.x] = d_SrcKey[0 + startSrcA + threadIdx.x];
        s_val[threadIdx.x] = d_SrcVal[0 + startSrcA + threadIdx.x];
        s_inf[threadIdx.x] = 0;
    }

    //Prepare for bitonic merge by inversing the ordering
    if (threadIdx.x < lenSrcB)
    {
        s_key[2 * SAMPLE_STRIDE - 1 - threadIdx.x] = d_SrcKey[stride + startSrcB + threadIdx.x];
        s_val[2 * SAMPLE_STRIDE - 1 - threadIdx.x] = d_SrcVal[stride + startSrcB + threadIdx.x];
        s_inf[2 * SAMPLE_STRIDE - 1 - threadIdx.x] = 0;
    }

    //"Extended" bitonic merge
    for (uint stride = SAMPLE_STRIDE; stride > 0; stride >>= 1)
    {
        __syncthreads();
        uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
        ComparatorExtended<sortDir>(
            s_key[pos +      0], s_val[pos +      0], s_inf[pos +      0],
            s_key[pos + stride], s_val[pos + stride], s_inf[pos + stride],
            sortDir
        );
    }

    //Store sorted data
    __syncthreads();
    d_DstKey += startDst;
    d_DstVal += startDst;

    if (threadIdx.x < lenSrcA)
    {
        d_DstKey[threadIdx.x] = s_key[threadIdx.x];
        d_DstVal[threadIdx.x] = s_val[threadIdx.x];
    }

    if (threadIdx.x < lenSrcB)
    {
        d_DstKey[lenSrcA + threadIdx.x] = s_key[lenSrcA + threadIdx.x];
        d_DstVal[lenSrcA + threadIdx.x] = s_val[lenSrcA + threadIdx.x];
    }
}
