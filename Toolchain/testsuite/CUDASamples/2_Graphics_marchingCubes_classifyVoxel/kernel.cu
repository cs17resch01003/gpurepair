#include "hip/hip_runtime.h"
//pass
//--gridDim=[256,1,1] --blockDim=[128,1,1]

#include "common.h"

__global__ void
classifyVoxel(uint *voxelVerts, uint *voxelOccupied, uchar *volume,
              uint3 gridSize, uint3 gridSizeShift, uint3 gridSizeMask, uint numVoxels,
              float3 voxelSize, float isoValue)
{
    uint blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
    uint i = __mul24(blockId, blockDim.x) + threadIdx.x;

    uint3 gridPos = calcGridPos(i, gridSizeShift, gridSizeMask);

    // read field values at neighbouring grid vertices
#if SAMPLE_VOLUME
    float field[8];
    field[0] = sampleVolume(volume, gridPos, gridSize);
    field[1] = sampleVolume(volume, gridPos + make_uint3(1, 0, 0), gridSize);
    field[2] = sampleVolume(volume, gridPos + make_uint3(1, 1, 0), gridSize);
    field[3] = sampleVolume(volume, gridPos + make_uint3(0, 1, 0), gridSize);
    field[4] = sampleVolume(volume, gridPos + make_uint3(0, 0, 1), gridSize);
    field[5] = sampleVolume(volume, gridPos + make_uint3(1, 0, 1), gridSize);
    field[6] = sampleVolume(volume, gridPos + make_uint3(1, 1, 1), gridSize);
    field[7] = sampleVolume(volume, gridPos + make_uint3(0, 1, 1), gridSize);
#else
    float3 p;
    p.x = -1.0f + (gridPos.x * voxelSize.x);
    p.y = -1.0f + (gridPos.y * voxelSize.y);
    p.z = -1.0f + (gridPos.z * voxelSize.z);

    float field[8];
    field[0] = fieldFunc(p);
    field[1] = fieldFunc(p + make_float3(voxelSize.x, 0, 0));
    field[2] = fieldFunc(p + make_float3(voxelSize.x, voxelSize.y, 0));
    field[3] = fieldFunc(p + make_float3(0, voxelSize.y, 0));
    field[4] = fieldFunc(p + make_float3(0, 0, voxelSize.z));
    field[5] = fieldFunc(p + make_float3(voxelSize.x, 0, voxelSize.z));
    field[6] = fieldFunc(p + make_float3(voxelSize.x, voxelSize.y, voxelSize.z));
    field[7] = fieldFunc(p + make_float3(0, voxelSize.y, voxelSize.z));
#endif

    // calculate flag indicating if each vertex is inside or outside isosurface
    uint cubeindex;
    cubeindex =  uint(field[0] < isoValue);
    cubeindex += uint(field[1] < isoValue)*2;
    cubeindex += uint(field[2] < isoValue)*4;
    cubeindex += uint(field[3] < isoValue)*8;
    cubeindex += uint(field[4] < isoValue)*16;
    cubeindex += uint(field[5] < isoValue)*32;
    cubeindex += uint(field[6] < isoValue)*64;
    cubeindex += uint(field[7] < isoValue)*128;

    // read number of vertices from texture
    uint numVerts = tex1Dfetch(numVertsTex, cubeindex);

    if (i < numVoxels)
    {
        voxelVerts[i] = numVerts;
        voxelOccupied[i] = (numVerts > 0);
    }
}
