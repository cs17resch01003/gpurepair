#include "hip/hip_runtime.h"
//pass
//--gridDim=[32,32] --blockDim=[16,16]

__device__ static __attribute__((always_inline)) int rgbToInt(float r, float g, float b);
__device__ static __attribute__((always_inline)) uchar4 getPixel(int x, int y);

#ifndef USE_TEXTURE_RGBA8UI
texture<float4, 2, hipReadModeElementType> inTex;
#else
texture<uchar4, 2, hipReadModeElementType> inTex;
#endif

// convert floating point rgb color to 8-bit integer
__device__ static __attribute__((always_inline)) int rgbToInt(float r, float g, float b)
{
    r = clamp(r, 0.0f, 255.0f);
    g = clamp(g, 0.0f, 255.0f);
    b = clamp(b, 0.0f, 255.0f);
    return (int(b)<<16) | (int(g)<<8) | int(r);
}

// get pixel from 2D image, with clamping to border
__device__ static __attribute__((always_inline)) uchar4 getPixel(int x, int y)
{
#ifndef USE_TEXTURE_RGBA8UI
    float4 res = tex2D(inTex, x, y);
    uchar4 ucres = make_uchar4(res.x*255.0f, res.y*255.0f, res.z*255.0f, res.w*255.0f);
#else
    uchar4 ucres = tex2D(inTex, x, y);
#endif
    return ucres;
}

// macros to make indexing shared memory easier
#define SMEM(X, Y) sdata[(Y)*tilew+(X)]

__global__ void
cudaProcess(unsigned int *g_odata, int imgw, int imgh,
            int tilew, int r, float threshold, float highlight)
{
    __requires(imgw == 512);
    __requires(tilew == 32);
    __requires(r == 8);
    extern __shared__ uchar4 sdata[];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bw = blockDim.x;
    int bh = blockDim.y;
    int x = blockIdx.x*bw + tx;
    int y = blockIdx.y*bh + ty;

#if 0
    uchar4 c4 = getPixel(x, y);
    g_odata[y*imgw+x] = rgbToInt(c4.z, c4.y, c4.x);
#else
    // copy tile to shared memory
    // center region
    SMEM(r + tx, r + ty) = getPixel(x, y);

    // borders
    if (threadIdx.x < r)
    {
        // left
        SMEM(tx, r + ty) = getPixel(x - r, y);
        // right
        SMEM(r + bw + tx, r + ty) = getPixel(x + bw, y);
    }

    if (threadIdx.y < r)
    {
        // top
        SMEM(r + tx, ty) = getPixel(x, y - r);
        // bottom
        SMEM(r + tx, r + bh + ty) = getPixel(x, y + bh);
    }

    // load corners
    if ((threadIdx.x < r) && (threadIdx.y < r))
    {
        // tl
        SMEM(tx, ty) = getPixel(x - r, y - r);
        // bl
        SMEM(tx, r + bh + ty) = getPixel(x - r, y + bh);
        // tr
        SMEM(r + bw + tx, ty) = getPixel(x + bh, y - r);
        // br
        SMEM(r + bw + tx, r + bh + ty) = getPixel(x + bw, y + bh);
    }

    // wait for loads to complete
    // __syncthreads();

    // perform convolution
    float rsum = 0.0f;
    float gsum = 0.0f;
    float bsum = 0.0f;
    float samples = 0.0f;

    for (int dy=-r; dy<=r; dy++)
    {
        for (int dx=-r; dx<=r; dx++)
        {
#if 0
            // try this to see the benefit of using shared memory
            uchar4 pixel = getPixel(x+dx, y+dy);
#else
            uchar4 pixel = SMEM(r+tx+dx, r+ty+dy);
#endif

            // only sum pixels within disc-shaped kernel
            float l = dx*dx + dy*dy;

            if (l <= r*r)
            {
                float r = float(pixel.x);
                float g = float(pixel.y);
                float b = float(pixel.z);
#if 1
                // brighten highlights
                float lum = (r + g + b) / (255*3);

                if (lum > threshold)
                {
                    r *= highlight;
                    g *= highlight;
                    b *= highlight;
                }

#endif
                rsum += r;
                gsum += g;
                bsum += b;
                samples += 1.0f;
            }
        }
    }

    rsum /= samples;
    gsum /= samples;
    bsum /= samples;
    // ABGR
    g_odata[y*imgw+x] = rgbToInt(rsum, gsum, bsum);
    //g_odata[y*imgw+x] = rgbToInt(x,y,0);
#endif
}
